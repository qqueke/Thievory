#include "sssp.cuh"
#include <iostream>

void SSSP32(string filePath, uint32 srcVertex, double memAdvise, uint32 nRuns) {
  ALGORITHM_TYPE algo = SSSP;
  CSR<uint32> *graph = new CSR<uint32>;
  graph->ReadInputFile(filePath, algo);
  graph->InitData(srcVertex);
  // Adjust this number of blocks in x dimension to be a multiple of the number
  // of SMS and acquire better load balancing
  int device = 0; // Selected device
  uint32 k =
      4; // Multiple of SMs to choose for the grid dimension (to be adjusted)

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint32 numSMs = prop.multiProcessorCount;

  dim3 staticGrid = dim3(k * numSMs, 1, 1);
  dim3 blockDim(THREADS_PER_BLOCK, 1, 1); // (x,y,z) = (THREADS_PER_BLOCK, 1, 1)

  hipStream_t staticStream, demandStream, frontierStream;

  GPUAssert(hipStreamCreate(&frontierStream));
  GPUAssert(hipStreamCreate(&staticStream));
  GPUAssert(hipStreamCreate(&demandStream));

  auto asyncFrontierPolicy = thrust::cuda::par_nosync.on(frontierStream);
  auto asyncStaticPolicy = thrust::cuda::par_nosync.on(staticStream);
  auto asyncDemandPolicy = thrust::cuda::par_nosync.on(demandStream);
  // auto syncPolicy  = thrust::cuda::par.on(staticStream);

  auto syncFrontierPolicy = thrust::cuda::par.on(frontierStream);
  auto syncStaticPolicy = thrust::cuda::par.on(staticStream);
  auto syncDemandPolicy = thrust::cuda::par.on(demandStream);

  TimeRecord<chrono::milliseconds> totalProcess("Total execution");

  // Removing static data
  hipMemset(graph->d_inStatic, 0, *(graph->numVertices) * sizeof(bool));

  std::cout << "Starting Traversals" << std::endl;
  for (int test = 0; test < nRuns; test++) {

    graph->ResetFrontierNValues();

    *(graph->frontierSize) = thrust::reduce(
        graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices), 0,
        thrust::plus<uint32>());

    totalProcess.startRecord();

    while (*(graph->frontierSize)) {

      setStaticNDemandFrontiers<<<staticGrid, blockDim, 0, frontierStream>>>(
          graph->numVertices, graph->d_frontier, graph->d_staticFrontier,
          graph->d_demandFrontier, graph->d_inStatic);

      hipStreamSynchronize(frontierStream);

      hipMemsetAsync(graph->d_frontier, 0,
                      *(graph->numVertices) * sizeof(*graph->d_frontier),
                      frontierStream);

      // Calculate the amount of active nodes in GPU memory
      *(graph->staticSize) =
          thrust::reduce(graph->thurstStaticFrontier,
                         graph->thurstStaticFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      // Calculate the amount of active vertices on-demand
      *(graph->demandSize) =
          thrust::reduce(graph->thurstDemandFrontier,
                         graph->thurstDemandFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*(graph->staticSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstStaticFrontier,
            graph->thurstStaticFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setStaticList<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->numVertices, graph->d_staticList, graph->d_staticFrontier,
            graph->d_prefixSum);

        hipStreamSynchronize(frontierStream);

        SSSP32_Static_Kernel<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->staticSize, graph->d_staticList, graph->d_offsets,
            graph->d_staticEdges, graph->d_staticWeights, graph->d_values,
            graph->d_frontier, graph->d_staticFrontier);
      }

      if (*(graph->demandSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstDemandFrontier,
            graph->thurstDemandFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setDemandList<<<staticGrid, blockDim, 0, demandStream>>>(
            graph->numVertices, graph->d_demandList, graph->d_demandFrontier,
            graph->d_prefixSum);

        uint32 numBlocks =
            (((*(graph->demandSize)) * WARP_SIZE + THREADS_PER_BLOCK) /
             THREADS_PER_BLOCK);
        dim3 gridDim(THREADS_PER_BLOCK,
                     (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

        hipStreamSynchronize(demandStream);

        SSSP32_Demand_Kernel<<<gridDim, blockDim, 0, demandStream>>>(
            graph->demandSize, graph->d_demandList, graph->d_values,
            graph->d_frontier, graph->h_edges, graph->h_weights,
            graph->d_offsets);
      }

      GPUAssert(hipDeviceSynchronize());

      *(graph->frontierSize) = thrust::reduce(
          graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices),
          0, thrust::plus<uint32>());
    }
  }

  totalProcess.endRecord();
  totalProcess.print();

  // We're gonna need to compare results now!!
  hipMemcpy(graph->h_values, graph->d_values,
             *(graph->numVertices) * sizeof(uint32), hipMemcpyDeviceToHost);

  graph->DumpValues();
  return;
}

void SSSP64(string filePath, uint32 srcVertex, double memAdvise, uint32 nRuns) {

  return;
}
