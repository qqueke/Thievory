#include "hip/hip_runtime.h"
#include "cc_kernels.cuh"

// Testar o mesmo metodo com warps que utilizamos pro demand
__global__ void
CC32_Static_Kernel(const uint32 *staticSize, const uint32 *d_staticList,
                   const uint64 *d_offsets, const uint32 *d_staticEdges,
                   uint32 *d_values, bool *d_frontier, const bool *d_inStatic) {
  // for (uint32 index = blockIdx.x * blockDim.x + threadIdx.x;
  //     index < *staticSize; index += blockDim.x * gridDim.x) {
  //  uint32 vertexId = d_staticList[index];

  //  // Pretty sure we can remove this but lets review it first
  //  // if (d_inStatic[vertexId])
  //  // {
  //  // CC specific
  //  uint32 sourceValue = d_values[vertexId];

  //  // Neighbors to access
  //  uint64 startNeighbor = d_offsets[vertexId];
  //  uint64 endNeighbor = d_offsets[vertexId + 1];

  //  for (uint64 i = startNeighbor; i < endNeighbor; i++) {
  //    uint32 neighborId = d_staticEdges[i];

  //    // If this new path has lower cost than the previous then change and add
  //    // the neighbor to the frontier
  //    if (sourceValue < d_values[neighborId]) {
  //      atomicMin(&d_values[neighborId], sourceValue);
  //      d_frontier[neighborId] = 1;
  //    }
  //  }
  //  // }
  //}

  // (Row) + (Column) + (Thread Offset)

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;
  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < *staticSize; warpIdx += numWarps) {
    const uint32 traverseIndex = warpIdx;
    uint32 vertexId = d_staticList[traverseIndex];

    uint32 sourceValue = d_values[vertexId];

    const uint64 start = d_offsets[vertexId];
    const uint64 end = d_offsets[vertexId + 1];

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      if (i >= start) {
        uint32 neighborId = d_staticEdges[i];

        // If this new path has lower cost than the previous then change and add
        // the neighbor to the frontier
        if (sourceValue < d_values[neighborId]) {
          atomicMin(&d_values[neighborId], sourceValue);
          d_frontier[neighborId] = 1;
        }
      }
    }
  }
}

__global__ void
CC64_Static_Kernel(const uint64 *staticSize, const uint64 *d_staticList,
                   const uint64 *d_offsets, const uint64 *d_staticEdges,
                   uint64 *d_values, bool *d_frontier, const bool *d_inStatic) {
  for (uint32 index = blockIdx.x * blockDim.x + threadIdx.x;
       index < *staticSize; index += blockDim.x * gridDim.x) {
    uint64 vertexId = d_staticList[index];

    // Pretty sure we can remove this but lets review it first
    if (d_inStatic[vertexId]) {
      // BFS specific
      uint64 sourceValue = d_values[vertexId];

      // Neighbors to access
      uint64 startNeighbor = d_offsets[vertexId];
      uint64 endNeighbor = d_offsets[vertexId + 1];

      for (uint64 i = startNeighbor; i < endNeighbor; i++) {
        uint64 neighborId = d_staticEdges[i];

        // If this new path has lower cost than the previous then change and add
        // the neighbor to the frontier
        if (sourceValue < d_values[neighborId]) {
          atomicMin(&d_values[neighborId], sourceValue);
          d_frontier[neighborId] = 1;
        }
      }
    }
  }
}

__global__ void CC32_Demand_Kernel(const uint32 *demandSize,
                                   const uint32 *d_demandList, uint32 *d_values,
                                   bool *d_frontier, const uint32 *h_edges,
                                   const uint64 *d_offsets) {
  // (Row) + (Column) + (Thread Offset)
  const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
                     blockDim.x * blockIdx.x + threadIdx.x;
  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < *demandSize; warpIdx += numWarps) {
    const uint32 traverseIndex = warpIdx;
    uint32 vertexId = d_demandList[traverseIndex];

    uint32 sourceValue = d_values[vertexId];

    const uint64 start = d_offsets[vertexId];
    const uint64 shiftStart = start & MEM_ALIGN_32;
    const uint64 end = d_offsets[vertexId + 1];

    for (uint64 i = shiftStart + laneIdx; i < end; i += WARP_SIZE) {
      if (i >= start) {
        uint32 neighborId = h_edges[i];

        // If this new path has lower cost than the previous then change and add
        // the neighbor to the frontier
        if (sourceValue < d_values[neighborId]) {
          atomicMin(&d_values[neighborId], sourceValue);
          d_frontier[neighborId] = 1;
        }
      }
    }
  }
}

__global__ void CC64_Demand_Kernel(const uint64 *demandSize,
                                   const uint64 *d_demandList, uint64 *d_values,
                                   bool *d_frontier, const uint64 *h_edges,
                                   const uint64 *d_offsets) {
  // (Row) + (Column) + (Thread Offset)
  const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
                     blockDim.x * blockIdx.x + threadIdx.x;
  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < *demandSize; warpIdx += numWarps) {
    const uint32 traverseIndex = warpIdx;
    uint64 vertexId = d_demandList[traverseIndex];

    // uint64 srcValue = d_values[id];
    uint64 sourceValue = d_values[vertexId];

    const uint64 start = d_offsets[vertexId];
    const uint64 shiftStart = start & MEM_ALIGN_64;
    const uint64 end = d_offsets[vertexId + 1];

    for (uint64 i = shiftStart + laneIdx; i < end; i += WARP_SIZE) {
      if (i >= start) {
        uint64 neighborId = h_edges[i];

        // If this new path has lower cost than the previous then change and add
        // the neighbor to the frontier
        if (sourceValue < d_values[neighborId]) {
          atomicMin(&d_values[neighborId], sourceValue);
          d_frontier[neighborId] = 1;
        }
      }
    }
  }
}

__global__ void CC32_Filter_Kernel(const uint32 *partitionList,
                                   uint32 *d_partitionsOffsets,
                                   uint32 *d_values, bool *d_frontier,
                                   const uint32 *d_filterEdges,
                                   const uint64 *d_offsets,
                                   bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }

  //  uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  //
  //  uint32 partition = partitionList[0];
  //  // uint32 touch;
  //  //  Start Edge
  //  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];
  //
  //  for (tid += d_partitionsOffsets[partition];
  //       tid < d_partitionsOffsets[partition + 1];
  //       tid += blockDim.x * gridDim.x) {
  //
  //
  //    if (!d_filterFrontier[tid])
  //      continue;
  //
  //    d_filterFrontier[tid] = 0;
  //
  //    uint32 sourceValue = d_values[tid];
  //
  //    const uint64 start = d_offsets[tid] - startEdge;
  //    const uint64 end = d_offsets[tid + 1] - startEdge;
  //
  //    for (uint64 i = start; i < end; i ++ ) {
  //      uint32 neighborId = d_filterEdges[i];
  //
  //      // If this new path has lower cost than the previous then change and
  //      add
  //      // the neighbor to the frontier
  //      if (sourceValue < d_values[neighborId]) {
  //        atomicMin(&d_values[neighborId], sourceValue);
  //        d_frontier[neighborId] = 1;
  //      }
  //    }
  //  }
}

__global__ void CC32_NeighborFilter_Kernel(const uint32 *partitionList,
                                           uint32 *d_partitionsOffsets,
                                           uint32 *d_values, bool *d_frontier,
                                           const uint32 *d_filterEdges,
                                           const uint64 *d_offsets,
                                           bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;
  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }
}

__global__ void CC32_Static_Filter_Kernel(const uint32 *partitionList,
                                          uint32 *d_partitionsOffsets,
                                          uint32 *d_values, bool *d_frontier,
                                          const uint32 *d_filterEdges,
                                          const uint64 *d_offsets,
                                          bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  // uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;
  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx];
    const uint64 end = d_offsets[warpIdx + 1];

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }
}
