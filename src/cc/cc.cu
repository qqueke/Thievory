#include "hip/hip_runtime.h"
#include "cc.cuh"
#include <iostream>
#include <ostream>

#include <numa.h>
#include <queue>
#include <vector>

#define N_FILTER_STREAMS2 128

__global__ void CalculateCostNSplitFrontiers15(const uint32 *demandSize,
                                               uint32 *d_values,
                                               bool *d_frontier,
                                               const uint32 *h_edges,
                                               const uint64 *d_offsets) {
  // (Row) + (Column) + (Thread Offset)
  const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
                     blockDim.x * blockIdx.x + threadIdx.x;
  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < *demandSize; warpIdx += numWarps) {
    // if (!d_frontier[warpIdx]) {
    //   continue;
    // }
    uint32 touch;

    uint32 sourceValue = d_values[warpIdx];
    const uint64 start = d_offsets[warpIdx];
    const uint64 shiftStart = start & MEM_ALIGN_32;
    const uint64 end = d_offsets[warpIdx + 1];

    for (uint64 i = shiftStart + laneIdx; i < end; i += WARP_SIZE) {
      if (i >= start) {

        // uint32 neighborId = h_edges[i];
        //
        // if (sourceValue < d_values[neighborId]) {
        //   // atomicMin(&d_values[neighborId], sourceValue);
        //   //  d_frontier[neighborId] = 1;
        //   d_values[neighborId] = sourceValue;
        // }
        touch = h_edges[i];
        d_values[touch] = sourceValue;
      }
    }
  }
}

__global__ void CalculateCostNSplitFrontiers14(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets) {
  // (Row) + (Column) + (Thread Offset)
  const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
                     blockDim.x * blockIdx.x + threadIdx.x;
  uint32 warpIdx = tid >> 10;
  const uint32 laneIdx = tid & ((1 << 10) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / 1024;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < N_FILTER_STREAMS2; warpIdx += numWarps) {

    uint32 partition = partitionList[warpIdx];

    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]
    uint32 touch;
    // Start Edge
    uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

    // End Edge
    uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

    uint32 edgeCount = endEdge - startEdge;

    //  if (!d_frontier[warpIdx]) {
    //    continue;
    //  }

    uint32 sourceValue = d_values[d_partitionsOffsets[partition]];

    const uint64 start = warpIdx * EDGES_IN_PARTITION;
    const uint64 end = start + edgeCount;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {

      // d_values[warpIdx] += h_edges[warpIdx];

      touch = d_filterEdges[i + warpIdx * EDGES_IN_PARTITION];
      d_values[touch] = sourceValue;
    }
  }
}

__global__ void CalculateCostNSplitFrontiers11(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      touch = d_filterEdges[i];
      d_values[touch] = sourceValue;
    }
  }
}

__global__ void CalculateCostNSplitFrontiers16(const uint32 *demandSize,
                                               uint32 *d_values,
                                               bool *d_frontier,
                                               const uint32 *d_edges,
                                               const uint64 *d_offsets) {
  // (Row) + (Column) + (Thread Offset)
  const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
                     blockDim.x * blockIdx.x + threadIdx.x;
  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < *demandSize; warpIdx += numWarps) {
    // if (!d_frontier[warpIdx]) {
    //   continue;
    // }
    uint32 touch;

    uint32 sourceValue = d_values[warpIdx];
    const uint64 start = d_offsets[warpIdx];
    const uint64 end = d_offsets[warpIdx + 1];

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {

      // uint32 neighborId = h_edges[i];
      //
      // if (sourceValue < d_values[neighborId]) {
      //   // atomicMin(&d_values[neighborId], sourceValue);
      //   //  d_frontier[neighborId] = 1;
      //   d_values[neighborId] = sourceValue;
      // }
      touch = d_edges[i];
      d_values[touch] = sourceValue;
    }
  }
}

__global__ void CalculateCostNSplitFrontiers17(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets) {
  // (Row) + (Column) + (Thread Offset)
  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (; warpIdx < N_FILTER_STREAMS2; warpIdx += numWarps) {

    uint32 partition = partitionList[warpIdx];

    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]
    uint32 touch;

    // Start Edge
    uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

    // End Edge
    uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

    uint32 edgeCount = endEdge - startEdge;

    //  if (!d_frontier[warpIdx]) {
    //    continue;
    //  }

    uint32 sourceValue = d_values[d_partitionsOffsets[partition]];

    const uint64 start = warpIdx * EDGES_IN_PARTITION;
    const uint64 end = start + edgeCount;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {

      // d_values[warpIdx] += h_edges[warpIdx];

      touch = d_filterEdges[i + warpIdx * EDGES_IN_PARTITION];
      d_values[touch] = sourceValue;
    }
  }
}

__global__ void CalculateCostNSplitFrontiers20(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets,
    bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;

  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }

  //  uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  //
  //  uint32 partition = partitionList[0];
  //  // uint32 touch;
  //  //  Start Edge
  //  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];
  //
  //  for (tid += d_partitionsOffsets[partition];
  //       tid < d_partitionsOffsets[partition + 1];
  //       tid += blockDim.x * gridDim.x) {
  //
  //
  //    if (!d_filterFrontier[tid])
  //      continue;
  //
  //    d_filterFrontier[tid] = 0;
  //
  //    uint32 sourceValue = d_values[tid];
  //
  //    const uint64 start = d_offsets[tid] - startEdge;
  //    const uint64 end = d_offsets[tid + 1] - startEdge;
  //
  //    for (uint64 i = start; i < end; i ++ ) {
  //      uint32 neighborId = d_filterEdges[i];
  //
  //      // If this new path has lower cost than the previous then change and
  //      add
  //      // the neighbor to the frontier
  //      if (sourceValue < d_values[neighborId]) {
  //        atomicMin(&d_values[neighborId], sourceValue);
  //        d_frontier[neighborId] = 1;
  //      }
  //    }
  //  }
}

__global__ void CalculateCostNSplitFrontiers21(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets,
    bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;
  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }
}

__global__ void CalculateCostNSplitFrontiers22(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets,
    bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;
  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }
}
__global__ void CalculateCostNSplitFrontiers23(
    const uint32 *partitionList, uint32 *d_partitionsOffsets, uint32 *d_values,
    bool *d_frontier, const uint32 *d_filterEdges, const uint64 *d_offsets,
    bool *d_filterFrontier) {

  const uint32 tid = blockIdx.x * blockDim.x + threadIdx.x;
  // const uint32 tid = blockDim.x * THREADS_PER_BLOCK * blockIdx.y +
  //                    blockDim.x * blockIdx.x + threadIdx.x;

  uint32 warpIdx = tid >> WARP_SHIFT;
  const uint32 laneIdx = tid & ((1 << WARP_SHIFT) - 1);
  const uint32 numWarps = gridDim.x * gridDim.y * THREADS_PER_BLOCK / WARP_SIZE;

  uint32 partition = partitionList[0];
  // uint32 touch;
  //  Start Edge
  uint32 startEdge = d_offsets[d_partitionsOffsets[partition]];

  // End Edge
  // uint32 endEdge = d_offsets[d_partitionsOffsets[partition + 1]];

  // uint32 edgeCount = endEdge - startEdge;
  // Grid-Stride loop using Warp ID makes it easier to calculate with the .y
  // dimension
  for (warpIdx += d_partitionsOffsets[partition];
       warpIdx < d_partitionsOffsets[partition + 1]; warpIdx += numWarps) {
    // Start offset
    // d_partitionsOffsets[partition]

    // End offset
    // d_partitionsOffsets[partition + 1]

    if (!d_filterFrontier[warpIdx])
      continue;

    d_filterFrontier[warpIdx] = 0;

    uint32 sourceValue = d_values[warpIdx];

    const uint64 start = d_offsets[warpIdx] - startEdge;
    const uint64 end = d_offsets[warpIdx + 1] - startEdge;

    for (uint64 i = start + laneIdx; i < end; i += WARP_SIZE) {
      uint32 neighborId = d_filterEdges[i];

      // If this new path has lower cost than the previous then change and add
      // the neighbor to the frontier
      if (sourceValue < d_values[neighborId]) {
        atomicMin(&d_values[neighborId], sourceValue);
        d_frontier[neighborId] = 1;
      }
    }
  }
}

// #define ZC

void CC32(string filePath, double memAdvise, uint32 nRuns) {
  numa_run_on_node(0);
  ALGORITHM_TYPE algo = CC;
  CSR<uint32> *graph = new CSR<uint32>;
  graph->ReadInputFile(filePath, algo);
  graph->InitData(0);
  // Adjust this number of blocks in x dimension to be a multiple of the number
  // of SMS and acquire better load balancing
  int device = 0; // Selected device
  uint32 k =
      4; // Multiple of SMs to choose for the grid dimension (to be adjusted)

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint32 numSMs = prop.multiProcessorCount;

  dim3 staticGrid = dim3(k * numSMs, 1, 1);
  dim3 blockDim(THREADS_PER_BLOCK, 1, 1); // (x,y,z) = (THREADS_PER_BLOCK, 1, 1)

  hipStream_t staticStream, demandStream, frontierStream, demandStream1,
      demandStream2, demandStream3;

  GPUAssert(hipStreamCreate(&frontierStream));
  GPUAssert(hipStreamCreate(&staticStream));
  GPUAssert(hipStreamCreate(&demandStream));

  GPUAssert(hipStreamCreate(&demandStream1));

  GPUAssert(hipStreamCreate(&demandStream2));
  GPUAssert(hipStreamCreate(&demandStream3));

  auto asyncFrontierPolicy = thrust::cuda::par_nosync.on(frontierStream);
  auto asyncStaticPolicy = thrust::cuda::par_nosync.on(staticStream);
  auto asyncDemandPolicy = thrust::cuda::par_nosync.on(demandStream);
  // auto syncPolicy  = thrust::cuda::par.on(staticStream);

  auto syncFrontierPolicy = thrust::cuda::par.on(frontierStream);
  auto syncStaticPolicy = thrust::cuda::par.on(staticStream);
  auto syncDemandPolicy = thrust::cuda::par.on(demandStream);

  TimeRecord<chrono::milliseconds> totalProcess("Total execution");

  TimeRecord<chrono::milliseconds> test0("Copy to GPU 0");
  TimeRecord<chrono::milliseconds> test1("Copy to GPU 1");
  TimeRecord<chrono::milliseconds> test2("Copy to GPU 2");
  TimeRecord<chrono::milliseconds> test3("Copy to GPU 3");

  TimeRecord<chrono::milliseconds> k0("Kernel GPU 0");
  TimeRecord<chrono::milliseconds> k1("Kernel GPU 1");
  TimeRecord<chrono::milliseconds> k2("Kernel GPU 2");
  TimeRecord<chrono::milliseconds> k3("Kernel GPU 3");

  hipStream_t streams[N_FILTER_STREAMS2];

  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++)
    GPUAssert(hipStreamCreate(&streams[i]));

  hipStream_t neighborStreams[N_FILTER_STREAMS2];

  hipStream_t neighborComputeStreams[N_FILTER_STREAMS2];
  hipSetDevice(1);
  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++) {
    GPUAssert(hipStreamCreate(&neighborStreams[i]));
  }
  hipSetDevice(0);

  hipStream_t neighborStreams2[N_FILTER_STREAMS2];

  hipStream_t neighborComputeStreams2[N_FILTER_STREAMS2];
  hipSetDevice(2);
  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++) {
    GPUAssert(hipStreamCreate(&neighborStreams2[i]));
  }
  hipSetDevice(0);

  hipStream_t neighborStreams3[N_FILTER_STREAMS2];

  hipStream_t neighborComputeStreams3[N_FILTER_STREAMS2];
  hipSetDevice(3);
  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++) {
    GPUAssert(hipStreamCreate(&neighborStreams3[i]));
  }
  hipSetDevice(0);

  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++) {

    GPUAssert(hipStreamCreate(&neighborComputeStreams[i]));
    GPUAssert(hipStreamCreate(&neighborComputeStreams2[i]));
    GPUAssert(hipStreamCreate(&neighborComputeStreams3[i]));
  }

  // Removing static data
  // hipMemset(graph->d_inStatic, 0, *(graph->numVertices) * sizeof(bool));

  uint32 totalParts = 0;
  uint32 nGPUs = 4;
  GPUAssert(
      hipDeviceEnablePeerAccess(1, 0)); // Enable peer access with device 0

  GPUAssert(hipDeviceEnablePeerAccess(2, 0));

  GPUAssert(hipDeviceEnablePeerAccess(3, 0));

  graph->h_edges2 =
      (uint32 *)numa_alloc_onnode(graph->numEdges * sizeof(uint32), 1);

  hipHostRegister(graph->h_edges2, graph->numEdges * sizeof(uint32),
                   hipHostRegisterDefault);

  hipMemcpy(graph->h_edges2, graph->h_edges,
             graph->numEdges * sizeof(*graph->h_edges2), hipMemcpyHostToHost);

  hipDeviceSynchronize();

  bool *d_filterList; // Edges in host memory

  hipHostAlloc((void **)&d_filterList,
                *graph->numPartitions * sizeof(*d_filterList),
                hipHostMallocMapped);

  hipMemset(d_filterList, 0, *(graph->numPartitions) * sizeof(bool));

  hipDeviceSynchronize();
  std::cout << "Starting Traversals" << std::endl;
  for (int test = 0; test < nRuns; test++) {

    graph->ResetFrontierNValues();

    *(graph->frontierSize) = thrust::reduce(
        graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices), 0,
        thrust::plus<uint32>());

    totalProcess.startRecord();

    while (*(graph->frontierSize)) {

      //  std::cout << "Frontier size: " << *graph->frontierSize << std::endl;
      setStaticNDemandFrontiers<<<staticGrid, blockDim, 0, frontierStream>>>(
          graph->numVertices, graph->d_frontier, graph->d_staticFrontier,
          graph->d_demandFrontier, graph->d_inStatic);

      hipStreamSynchronize(frontierStream);

      hipMemsetAsync(graph->d_frontier, 0,
                      *(graph->numVertices) * sizeof(*graph->d_frontier),
                      frontierStream);

      // Calculate the amount of active nodes in GPU memory
      *(graph->staticSize) =
          thrust::reduce(graph->thurstStaticFrontier,
                         graph->thurstStaticFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*graph->frontierSize > 20000000) {
        CalculateCostNSplitFrontiers<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        CalculateCostNSplitFrontiers2<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        CalculateCostNSplitFrontiers3<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        hipStreamSynchronize(demandStream);

        // CalculateCostNSplitFrontiers4<uint32>
        //     <<<staticGrid, blockDim, 0, demandStream>>>(
        //         graph->numPartitions, graph->d_partitionCost, d_filterList);

        // hipDeviceSynchronize();

        // uint32 nPP = 0;
        // for (uint32 i = 0; i < *graph->numPartitions; i++) {
        //   if (d_filterList[i])
        //     nPP++;
        // }
        // std::cout << "Num partitions TO BE PROCESSED ACCORDING TO GPU: " <<
        // nPP
        //           << std::endl;

        // hipMemset(d_filterList, 0, *(graph->numPartitions) * sizeof(bool));

        // hipDeviceSynchronize();

        hipMemcpyAsync(graph->h_partitionCost, graph->d_partitionCost,
                        *graph->numPartitions * sizeof(*graph->h_partitionCost),
                        hipMemcpyDeviceToHost, streams[0]);

        // Verify this if we need sync here
        hipMemsetAsync(graph->d_partitionCost, 0,
                        *graph->numPartitions * sizeof(*graph->d_partitionCost),
                        streams[0]);
      }

      // Calculate the amount of active vertices on-demand
      *(graph->demandSize) =
          thrust::reduce(graph->thurstDemandFrontier,
                         graph->thurstDemandFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*(graph->staticSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstStaticFrontier,
            graph->thurstStaticFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setStaticList<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->numVertices, graph->d_staticList, graph->d_staticFrontier,
            graph->d_prefixSum);

        hipStreamSynchronize(frontierStream);

        CC32_Static_Kernel<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->staticSize, graph->d_staticList, graph->d_offsets,
            graph->d_staticEdges, graph->d_values, graph->d_frontier,
            graph->d_inStatic);
      }

      if (*(graph->demandSize) > 0) {
        thrust::exclusive_scan(
            graph->thurstDemandFrontier,
            graph->thurstDemandFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setDemandList<<<staticGrid, blockDim, 0, demandStream>>>(
            graph->numVertices, graph->d_demandList, graph->d_demandFrontier,
            graph->d_prefixSum);

        uint32 numBlocks =
            (((*(graph->demandSize)) * WARP_SIZE + THREADS_PER_BLOCK) /
             THREADS_PER_BLOCK);
        dim3 gridDim(THREADS_PER_BLOCK,
                     (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

        hipStreamSynchronize(frontierStream);

        // hipStreamSynchronize(staticStream); // Not sure

        CC32_Demand_Kernel<<<gridDim, blockDim, 0, demandStream>>>(
            graph->demandSize, graph->d_demandList, graph->d_values,
            graph->d_frontier, graph->h_edges, graph->d_offsets);
      }

      // if (*graph->frontierSize > 20000000) {

      //  hipMemcpy(graph->d_demandFrontier, graph->d_filterFrontier,
      //             *graph->numVertices * sizeof(*graph->d_filterFrontier),
      //             hipMemcpyDeviceToDevice);

      //  hipDeviceSynchronize();

      //  *(graph->demandSize) =
      //      thrust::reduce(graph->thurstDemandFrontier,
      //                     graph->thurstDemandFrontier +
      //                     *(graph->numVertices), 0, thrust::plus<uint32>());

      //  hipDeviceSynchronize();
      //   if (*(graph->demandSize) > 0) {
      //     thrust::exclusive_scan(
      //         graph->thurstDemandFrontier,
      //         graph->thurstDemandFrontier + *(graph->numVertices),
      //         graph->thurstPrefixSum, 0, thrust::plus<uint32>());

      //    setDemandList<<<staticGrid, blockDim, 0, demandStream>>>(
      //        graph->numVertices, graph->d_demandList,
      //        graph->d_demandFrontier, graph->d_prefixSum);

      //    uint32 numBlocks =
      //        (((*(graph->demandSize)) * WARP_SIZE + THREADS_PER_BLOCK) /
      //         THREADS_PER_BLOCK);
      //    dim3 gridDim(THREADS_PER_BLOCK,
      //                 (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

      //    hipStreamSynchronize(frontierStream);
      //    hipStreamSynchronize(staticStream);

      //    CC32_Demand_Kernel<<<gridDim, blockDim, 0, demandStream>>>(
      //        graph->demandSize, graph->d_demandList, graph->d_values,
      //        graph->d_frontier, graph->h_edges, graph->d_offsets);
      //  }
      //}
      if (*graph->frontierSize > 20000000) {

        uint32 numParts = 0;
        uint32 numPartsNGPU = 0;

        uint32 numPartsNGPU0 = 0;
        uint32 numPartsNGPU1 = 0;
        uint32 numPartsNGPU2 = 0;
        uint32 numPartsNGPU3 = 0;

        uint32 teoNumParts = 0;

        std::queue<uint32> targetGPUWorking;
        std::queue<uint32> targetGPUQueue;

        std::queue<uint32> neighborGPUQueue;

        std::queue<uint32> neighborGPUQueue2;

        std::queue<uint32> neighborGPUQueue3;

        std::vector<std::queue<uint32>> neighborGPUQueues;

        std::vector<uint32> partitionList;

        hipStreamSynchronize(streams[0]);

        for (uint32 partition = 0; partition < *graph->numPartitions;
             partition++) {

          if (graph->h_partitionCost[partition] <= FILTER_THRESHOLD)
            continue;

          partitionList.push_back(partition);
        }

        hipStreamSynchronize(frontierStream);
        // hipStreamSynchronize(staticStream); // Not sure
        // hipStreamSynchronize(demandStream); // Not sure

        for (uint32 index = 0; index < partitionList.size(); index++) {

          uint32 partition = partitionList[index];

          numParts++;

          // Partition edge start
          uint32 start =
              graph->h_offsets[graph->h_partitionsOffsets[partition]];

          uint32 partitionSize =
              graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
              start;

          // uint32 stream = partition % N_FILTER_STREAMS2;

          uint32 stream = (index / nGPUs) % N_FILTER_STREAMS2;

          graph->h_partitionList[stream] = partition;

          hipStreamSynchronize(streams[stream]);

          // while (!targetGPUWorking.empty()) {
          //   uint32 w = targetGPUWorking.front();
          //   hipStreamSynchronize(streams[w]);
          //   targetGPUWorking.pop();
          // }

          // hipDeviceSynchronize();
          test0.startRecord();
          hipMemcpyAsync(graph->d_filterEdges[stream], &graph->h_edges[start],
                          partitionSize * sizeof(*graph->h_edges),
                          hipMemcpyHostToDevice, streams[stream]);

          hipMemcpyAsync(&graph->d_partitionList[stream],
                          &graph->h_partitionList[stream],
                          sizeof(*graph->h_partitionList),
                          hipMemcpyHostToDevice, streams[stream]);

          //  hipDeviceSynchronize();
          test0.endRecord();

          targetGPUQueue.push(stream);

          if (index + 1 < partitionList.size()) {
            index++;
            partition = partitionList[index];

            teoNumParts++;

            // Partition edge start
            uint32 neighborStart =
                graph->h_offsets[graph->h_partitionsOffsets[partition]];

            // Partition edge ends
            //  uint32 end =
            //      graph->h_offsets[graph->h_partitionsOffsets[partition +1]];

            uint32 neighborPartitionSize =
                graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
                neighborStart;

            // uint32 stream = partition % N_FILTER_STREAMS2;

            uint32 neighborStream = (index / nGPUs) % N_FILTER_STREAMS2;

            graph->h_neighborPartitionList[neighborStream] = partition;

            hipStreamSynchronize(neighborComputeStreams[neighborStream]);
            hipSetDevice(1);

            //   hipDeviceSynchronize();
            test1.startRecord();
            hipMemcpyAsync(graph->d_neighborFilterEdges[neighborStream],
                            graph->h_edges + neighborStart,
                            neighborPartitionSize * sizeof(*graph->h_edges),
                            hipMemcpyHostToDevice,
                            neighborStreams[neighborStream]);

            hipMemcpyAsync(&graph->d_neighborPartitionList[neighborStream],
                            graph->h_neighborPartitionList + neighborStream,
                            sizeof(*graph->h_neighborPartitionList),
                            hipMemcpyHostToDevice,
                            neighborStreams[neighborStream]);
            //  hipDeviceSynchronize();
            test1.endRecord();

            hipSetDevice(0);

            neighborGPUQueue.push(neighborStream);
          }

          if (index + 1 < partitionList.size()) {
            index++;
            partition = partitionList[index];

            teoNumParts++;

            // Partition edge start
            uint32 neighborStart2 =
                graph->h_offsets[graph->h_partitionsOffsets[partition]];

            // Partition edge ends
            //  uint32 end =
            //      graph->h_offsets[graph->h_partitionsOffsets[partition +
            // 1]];

            uint32 neighborPartitionSize2 =
                graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
                neighborStart2;

            // uint32 stream = partition % N_FILTER_STREAMS2;

            uint32 neighborStream2 = (index / nGPUs) % N_FILTER_STREAMS2;
            // std::cout << "Neighbor stream 2:" << neighborStream2 <<
            // std::endl;
            graph->h_neighborPartitionList2[neighborStream2] = partition;

            hipStreamSynchronize(neighborComputeStreams2[neighborStream2]);
            hipSetDevice(2);
            // hipDeviceSynchronize();

            //   hipDeviceSynchronize();
            test2.startRecord();
            //  numa_run_on_node(1);
            hipMemcpyAsync(graph->d_neighborFilterEdges2[neighborStream2],
                            graph->h_edges2 + neighborStart2,
                            neighborPartitionSize2 * sizeof(*graph->h_edges),
                            hipMemcpyHostToDevice,
                            neighborStreams2[neighborStream2]);

            hipMemcpyAsync(&graph->d_neighborPartitionList2[neighborStream2],
                            graph->h_neighborPartitionList2 + neighborStream2,
                            sizeof(*graph->h_neighborPartitionList2),
                            hipMemcpyHostToDevice,
                            neighborStreams2[neighborStream2]);

            //    hipDeviceSynchronize();
            // numa_run_on_node(0);
            //     hipDeviceSynchronize();
            test2.endRecord();
            hipSetDevice(0);

            neighborGPUQueue2.push(neighborStream2);
          }

          if (index + 1 < partitionList.size()) {
            index++;
            partition = partitionList[index];

            teoNumParts++;

            // Partition edge start
            uint32 neighborStart3 =
                graph->h_offsets[graph->h_partitionsOffsets[partition]];

            // Partition edge ends
            //  uint32 end =
            //      graph->h_offsets[graph->h_partitionsOffsets[partition +
            // 1]];

            uint32 neighborPartitionSize3 =
                graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
                neighborStart3;

            // uint32 stream = partition % N_FILTER_STREAMS2;

            uint32 neighborStream3 = (index / nGPUs) % N_FILTER_STREAMS2;

            graph->h_neighborPartitionList3[neighborStream3] = partition;

            hipStreamSynchronize(neighborComputeStreams3[neighborStream3]);
            hipSetDevice(3);
            //   hipDeviceSynchronize();
            test3.startRecord();

            //  numa_run_on_node(1);
            hipMemcpyAsync(graph->d_neighborFilterEdges3[neighborStream3],
                            graph->h_edges2 + neighborStart3,
                            neighborPartitionSize3 * sizeof(*graph->h_edges),
                            hipMemcpyHostToDevice,
                            neighborStreams3[neighborStream3]);

            hipMemcpyAsync(&graph->d_neighborPartitionList3[neighborStream3],
                            graph->h_neighborPartitionList3 + neighborStream3,
                            sizeof(*graph->h_neighborPartitionList3),
                            hipMemcpyHostToDevice,
                            neighborStreams3[neighborStream3]);

            //  hipDeviceSynchronize();
            //  numa_run_on_node(0);
            test3.endRecord();

            // numa_run_on_node(0);
            hipSetDevice(0);

            neighborGPUQueue3.push(neighborStream3);
          }

          bool change = false;
          while (!targetGPUQueue.empty()) {
            uint32 i = targetGPUQueue.front();

            hipError_t streamStatus = hipStreamQuery(streams[i]);

            if (streamStatus == hipErrorNotReady) {
              if (targetGPUQueue.size() < N_FILTER_STREAMS2)
                continue;
              else
                hipStreamSynchronize(streams[i]);
            }

            change = true;
            numPartsNGPU0++;
            targetGPUQueue.pop();
            targetGPUWorking.push(i);
            // targetGPUWorking.push(i);
            //   hipDeviceSynchronize();
            k0.startRecord();
            CalculateCostNSplitFrontiers20<<<staticGrid, blockDim, 0,
                                             streams[i]>>>(
                &graph->d_partitionList[i], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier, graph->d_filterEdges[i],
                graph->d_offsets, graph->d_filterFrontier);
            //       hipDeviceSynchronize();
            k0.endRecord();
          }

          // if (change) {
          //   while (!targetGPUWorking.empty()) {
          //     uint32 w = targetGPUWorking.front();
          //     hipStreamSynchronize(streams[w]);
          //     targetGPUWorking.pop();
          //   }
          // }

          // hipDeviceSynchronize();
          // if (change) {
          //   hipDeviceSynchronize();
          //   // hipStreamSynchronize(demandStream);
          //   //  Verify if there are GPU streams running
          //   while (!targetGPUWorking.empty()) {
          //     uint32 w = targetGPUWorking.front();
          //     hipStreamSynchronize(streams[w]);
          //     targetGPUWorking.pop();
          //   }
          // }

          while (!neighborGPUQueue.empty()) {
            uint32 i = neighborGPUQueue.front();

            hipSetDevice(1);
            hipError_t streamStatus = hipStreamQuery(neighborStreams[i]);

            if (streamStatus == hipErrorNotReady) {
              if (neighborGPUQueue.size() < N_FILTER_STREAMS2) {
                hipSetDevice(0);
                continue;
              } else
                hipStreamSynchronize(neighborStreams[i]);
            }

            hipSetDevice(0);
            neighborGPUQueue.pop();
            numPartsNGPU++;
            numPartsNGPU1++;

            // while (!targetGPUWorking.empty()) {
            //   uint32 w = targetGPUWorking.front();
            //   hipStreamSynchronize(streams[w]);
            //   targetGPUWorking.pop();
            // }

            // demandStream1
            //  hipDeviceSynchronize();
            k1.startRecord();
            CalculateCostNSplitFrontiers21<<<staticGrid, blockDim, 0,
                                             neighborComputeStreams[i]>>>(
                &graph->d_neighborPartitionList[i], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_neighborFilterEdges[i], graph->d_offsets,
                graph->d_filterFrontier);
            //   hipDeviceSynchronize();
            k1.endRecord();
          }

          // hipDeviceSynchronize();
          while (!neighborGPUQueue2.empty()) {
            uint32 i = neighborGPUQueue2.front();

            hipSetDevice(2);
            hipError_t streamStatus = hipStreamQuery(neighborStreams2[i]);

            if (streamStatus == hipErrorNotReady) {
              if (neighborGPUQueue2.size() < N_FILTER_STREAMS2) {
                hipSetDevice(0);
                continue;
              } else
                hipStreamSynchronize(neighborStreams2[i]);
            }

            hipSetDevice(0);
            neighborGPUQueue2.pop();
            numPartsNGPU++;
            numPartsNGPU2++;

            // while (!targetGPUWorking.empty()) {
            //   uint32 w = targetGPUWorking.front();
            //   hipStreamSynchronize(streams[w]);
            //   targetGPUWorking.pop();
            // }

            // demandStream2
            //  hipDeviceSynchronize();
            k2.startRecord();
            CalculateCostNSplitFrontiers22<<<staticGrid, blockDim, 0,
                                             neighborComputeStreams2[i]>>>(
                &graph->d_neighborPartitionList2[i], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_neighborFilterEdges2[i], graph->d_offsets,
                graph->d_filterFrontier);
            //  hipDeviceSynchronize();
            k2.endRecord();
          }

          // hipDeviceSynchronize();
          while (!neighborGPUQueue3.empty()) {
            uint32 i = neighborGPUQueue3.front();

            hipSetDevice(3);
            hipError_t streamStatus = hipStreamQuery(neighborStreams3[i]);

            if (streamStatus == hipErrorNotReady) {
              if (neighborGPUQueue3.size() < N_FILTER_STREAMS2) {
                hipSetDevice(0);
                continue;
              } else
                hipStreamSynchronize(neighborStreams3[i]);
            }

            hipSetDevice(0);
            neighborGPUQueue3.pop();
            numPartsNGPU++;
            numPartsNGPU3++;

            // while (!targetGPUWorking.empty()) {
            //   uint32 w = targetGPUWorking.front();
            //   hipStreamSynchronize(streams[w]);
            //   targetGPUWorking.pop();
            // }
            //    hipDeviceSynchronize();
            //  demandStream3
            k3.startRecord();
            CalculateCostNSplitFrontiers23<<<staticGrid, blockDim, 0,
                                             neighborComputeStreams3[i]>>>(
                &graph->d_neighborPartitionList3[i], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_neighborFilterEdges3[i], graph->d_offsets,
                graph->d_filterFrontier);
            //   hipDeviceSynchronize();
            k3.endRecord();
          }

          // hipDeviceSynchronize();
        }

        // hipDeviceSynchronize();
        bool change = false;
        while (!targetGPUQueue.empty()) {

          uint32 i = targetGPUQueue.front();

          hipStreamSynchronize(streams[i]);
          change = true;
          numPartsNGPU0++;
          targetGPUQueue.pop();

          // demandStream
          //    hipDeviceSynchronize();
          k0.startRecord();
          CalculateCostNSplitFrontiers20<<<staticGrid, blockDim, 0,
                                           streams[i]>>>(
              &graph->d_partitionList[i], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier, graph->d_filterEdges[i],
              graph->d_offsets, graph->d_filterFrontier);
          //   hipDeviceSynchronize();
          k0.endRecord();
        }

        // if (change)
        //   hipDeviceSynchronize();

        while (!neighborGPUQueue.empty()) {
          uint32 i = neighborGPUQueue.front();

          hipSetDevice(1);
          hipError_t streamStatus = hipStreamQuery(neighborStreams[i]);

          if (streamStatus == hipErrorNotReady) {
            if (neighborGPUQueue.size() < N_FILTER_STREAMS2) {
              hipSetDevice(0);
              continue;
            } else
              hipStreamSynchronize(neighborStreams[i]);
          }

          // Verify if there are GPU streams running
          //  while (!targetGPUWorking.empty()) {
          //    uint32 w = targetGPUWorking.front();
          //    hipStreamSynchronize(streams[w]);
          //    targetGPUWorking.pop();
          //  }

          hipSetDevice(0);
          neighborGPUQueue.pop();
          numPartsNGPU++;
          numPartsNGPU1++;
          // demandStream1
          //   hipDeviceSynchronize();
          k1.startRecord();
          CalculateCostNSplitFrontiers21<<<staticGrid, blockDim, 0,
                                           neighborComputeStreams[i]>>>(
              &graph->d_neighborPartitionList[i], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier,
              graph->d_neighborFilterEdges[i], graph->d_offsets,
              graph->d_filterFrontier);
          //    hipDeviceSynchronize();
          k1.endRecord();
        }

        // hipDeviceSynchronize();
        while (!neighborGPUQueue2.empty()) {

          uint32 i = neighborGPUQueue2.front();

          hipSetDevice(2);

          hipStreamSynchronize(neighborStreams2[i]);

          hipSetDevice(0);
          neighborGPUQueue2.pop();
          numPartsNGPU++;

          numPartsNGPU2++;
          //  hipDeviceSynchronize();
          k2.startRecord();
          CalculateCostNSplitFrontiers22<<<staticGrid, blockDim, 0,
                                           neighborComputeStreams2[i]>>>(
              &graph->d_neighborPartitionList2[i], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier,
              graph->d_neighborFilterEdges2[i], graph->d_offsets,
              graph->d_filterFrontier);
          //  hipDeviceSynchronize();
          k2.endRecord();
        }

        //  hipDeviceSynchronize();
        while (!neighborGPUQueue3.empty()) {

          uint32 i = neighborGPUQueue3.front();

          hipSetDevice(3);

          hipStreamSynchronize(neighborStreams3[i]);

          hipSetDevice(0);
          neighborGPUQueue3.pop();
          numPartsNGPU++;
          numPartsNGPU3++;
          //   hipDeviceSynchronize();
          k3.startRecord();
          CalculateCostNSplitFrontiers23<<<staticGrid, blockDim, 0,
                                           neighborComputeStreams3[i]>>>(
              &graph->d_neighborPartitionList3[i], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier,
              graph->d_neighborFilterEdges3[i], graph->d_offsets,
              graph->d_filterFrontier);
          //   hipDeviceSynchronize();
          k3.endRecord();
        }
        std::cout << "Partitions to be processed target GPU: " << numParts
                  << std::endl;
        std::cout << "Partitions processed in GPU 0: " << numPartsNGPU0
                  << std::endl;

        std::cout << "Partitions to be processed in neighbor GPUs: "
                  << teoNumParts << std::endl;

        std::cout << "Partitions processed in neighbor GPUs: " << numPartsNGPU
                  << std::endl;
        std::cout << "Partitions processed in neighbor GPU 1: " << numPartsNGPU1
                  << std::endl;
        std::cout << "Partitions processed in neighbor GPU 2: " << numPartsNGPU2
                  << std::endl;

        std::cout << "Partitions processed in neighbor GPU 3: " << numPartsNGPU3
                  << std::endl;

        totalParts += numParts + teoNumParts;
      }
      hipDeviceSynchronize();
      // GPUAssert(hipDeviceSynchronize());

      *(graph->frontierSize) = thrust::reduce(
          graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices),
          0, thrust::plus<uint32>());
    }
  }

  totalProcess.endRecord();
  totalProcess.print();

  test0.print();
  test1.print();
  test2.print();
  test3.print();

  k0.print();
  k1.print();
  k2.print();
  k3.print();

  uint64 MBytes = totalParts * 16;

  uint64 GBytes = MBytes >> 10;
  std::cout << "Total partitions in filter: " << totalParts << std::endl;

  std::cout << "Total amount of data sent with filter: " << MBytes << " MB"
            << std::endl;
  std::cout << "Total amount of data sent with filter: " << GBytes << " GB"
            << std::endl; // We're gonna need to compare results now!!
  hipMemcpy(graph->h_values, graph->d_values,
             *(graph->numVertices) * sizeof(uint32), hipMemcpyDeviceToHost);

  graph->DumpValues();
  return;
}

void CC64(string filePath, double memAdvise, uint32 nRuns) {
  ALGORITHM_TYPE algo = CC;
  CSR<uint32> *graph = new CSR<uint32>;
  graph->ReadInputFile(filePath, algo);
  graph->InitData(0);
  // Adjust this number of blocks in x dimension to be a multiple of the
  // number of SMS and acquire better load balancing
  int device = 0; // Selected device
  uint32 k =
      4; // Multiple of SMs to choose for the grid dimension (to be adjusted)

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint32 numSMs = prop.multiProcessorCount;

  dim3 staticGrid = dim3(k * numSMs, 1, 1);
  dim3 blockDim(THREADS_PER_BLOCK, 1,
                1); // (x,y,z) = (THREADS_PER_BLOCK, 1, 1)

  hipStream_t streams[16];

  for (uint32 i = 0; i < 16; i++)
    GPUAssert(hipStreamCreate(&streams[i]));

  TimeRecord<chrono::milliseconds> totalProcess("Total execution");

  // Removing static data
  // hipMemset(graph->d_inStatic, 0, *(graph->numVertices) * sizeof(bool));

  uint32 *d_edges;
  GPUAssert(hipMalloc(&d_edges, graph->numEdges * sizeof(*d_edges)));

  std::cout << "Starting Traversals" << std::endl;
  for (int test = 0; test < nRuns; test++) {

    graph->ResetFrontierNValues();

    totalProcess.startRecord();

#ifdef ZC
    uint32 numBlocks =
        (((*(graph->numVertices)) * WARP_SIZE + THREADS_PER_BLOCK) /
         THREADS_PER_BLOCK);
    dim3 gridDim(THREADS_PER_BLOCK,
                 (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

    CalculateCostNSplitFrontiers15<<<gridDim, blockDim, 0, streams[0]>>>(
        graph->numVertices, graph->d_values, graph->d_frontier, graph->h_edges,
        graph->d_offsets);
#else

    //   hipMemcpyAsync(d_edges, graph->h_edges, graph->numEdges *
    //   sizeof(*d_edges),
    //                   hipMemcpyHostToDevice, streams[0]);

    //   uint32 numBlocks =
    //       (((*(graph->numVertices)) * WARP_SIZE + THREADS_PER_BLOCK) /
    //        THREADS_PER_BLOCK);
    //   dim3 gridDim(THREADS_PER_BLOCK,
    //                (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

    //   CalculateCostNSplitFrontiers16<<<gridDim, blockDim, 0, streams[0]>>>(
    //       graph->numVertices, graph->d_values, graph->d_frontier, d_edges,
    //       graph->d_offsets);

    uint32 *d_partitionList0;
    uint32 *d_partitionList1;
    uint32 *d_partitionList2;
    uint32 *d_partitionList3;
    uint32 *d_partitionList4;
    uint32 *d_partitionList5;
    uint32 *d_partitionList6;
    uint32 *d_partitionList7;
    uint32 *d_partitionList8;
    uint32 *d_partitionList9;
    uint32 *d_partitionList10;
    uint32 *d_partitionList11;
    uint32 *d_partitionList12;
    uint32 *d_partitionList13;
    uint32 *d_partitionList14;
    uint32 *d_partitionList15;
    GPUAssert(hipMalloc(&d_partitionList0, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList1, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList2, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList3, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList4, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList5, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList6, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList7, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList8, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList9, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList10, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList11, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList12, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList13, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList14, sizeof(*d_partitionList0)));
    GPUAssert(hipMalloc(&d_partitionList15, sizeof(*d_partitionList0)));

    uint32 *d_filterEdges0;
    uint32 *d_filterEdges1;
    uint32 *d_filterEdges2;
    uint32 *d_filterEdges3;
    uint32 *d_filterEdges4;
    uint32 *d_filterEdges5;
    uint32 *d_filterEdges6;
    uint32 *d_filterEdges7;
    uint32 *d_filterEdges8;
    uint32 *d_filterEdges9;
    uint32 *d_filterEdges10;
    uint32 *d_filterEdges11;
    uint32 *d_filterEdges12;
    uint32 *d_filterEdges13;
    uint32 *d_filterEdges14;
    uint32 *d_filterEdges15;

    GPUAssert(hipMalloc(&d_filterEdges0,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges0)));
    GPUAssert(hipMalloc(&d_filterEdges1,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges1)));
    GPUAssert(hipMalloc(&d_filterEdges2,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges2)));
    GPUAssert(hipMalloc(&d_filterEdges3,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges3)));
    GPUAssert(hipMalloc(&d_filterEdges4,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges4)));
    GPUAssert(hipMalloc(&d_filterEdges5,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges5)));
    GPUAssert(hipMalloc(&d_filterEdges6,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges6)));
    GPUAssert(hipMalloc(&d_filterEdges7,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges7)));
    GPUAssert(hipMalloc(&d_filterEdges8,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges8)));
    GPUAssert(hipMalloc(&d_filterEdges9,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges9)));
    GPUAssert(hipMalloc(&d_filterEdges10,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges10)));
    GPUAssert(hipMalloc(&d_filterEdges11,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges11)));
    GPUAssert(hipMalloc(&d_filterEdges12,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges12)));
    GPUAssert(hipMalloc(&d_filterEdges13,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges13)));
    GPUAssert(hipMalloc(&d_filterEdges14,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges14)));
    GPUAssert(hipMalloc(&d_filterEdges15,
                         EDGES_IN_PARTITION * sizeof(*d_filterEdges15)));

    for (uint32 partition = 0; partition < *graph->numPartitions; partition++) {

      // Partition edge start
      uint32 start = graph->h_offsets[graph->h_partitionsOffsets[partition]];
      // Partition edge end
      uint32 end = graph->h_offsets[graph->h_partitionsOffsets[partition + 1]];

      uint32 partitionSize = end - start;

      uint32 stream = partition % N_FILTER_STREAMS2;

      graph->h_partitionList[stream] = partition;

      switch (stream) {
      case 0:
        hipMemcpyAsync(d_filterEdges0, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList0, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList0, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges0, graph->d_offsets);
        break;
      case 1:
        hipMemcpyAsync(d_filterEdges1, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList1, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList1, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges1, graph->d_offsets);
        break;
      case 2:
        hipMemcpyAsync(d_filterEdges2, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList2, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList2, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges2, graph->d_offsets);
        break;
      case 3:
        hipMemcpyAsync(d_filterEdges3, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList3, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList3, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges3, graph->d_offsets);
        break;
      case 4:
        hipMemcpyAsync(d_filterEdges4, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList4, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList4, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges4, graph->d_offsets);
        break;
      case 5:
        hipMemcpyAsync(d_filterEdges5, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList5, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList5, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges5, graph->d_offsets);
        break;
      case 6:
        hipMemcpyAsync(d_filterEdges6, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList6, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList6, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges6, graph->d_offsets);
        break;
      case 7:
        hipMemcpyAsync(d_filterEdges7, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList7, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList7, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges7, graph->d_offsets);
        break;
      case 8:
        hipMemcpyAsync(d_filterEdges8, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList8, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList8, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges8, graph->d_offsets);
        break;
      case 9:
        hipMemcpyAsync(d_filterEdges9, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList9, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList9, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges9, graph->d_offsets);
        break;
      case 10:
        hipMemcpyAsync(d_filterEdges10, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList10, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList10, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges10, graph->d_offsets);
        break;
      case 11:
        hipMemcpyAsync(d_filterEdges11, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList11, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList11, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges11, graph->d_offsets);
        break;
      case 12:
        hipMemcpyAsync(d_filterEdges12, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList12, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList12, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges12, graph->d_offsets);
        break;
      case 13:
        hipMemcpyAsync(d_filterEdges13, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList13, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList13, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges13, graph->d_offsets);
        break;
      case 14:
        hipMemcpyAsync(d_filterEdges14, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList14, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList14, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges14, graph->d_offsets);
        break;
      case 15:
        hipMemcpyAsync(d_filterEdges15, graph->h_edges + start,
                        partitionSize * sizeof(*graph->h_edges),
                        hipMemcpyHostToDevice, streams[stream]);

        hipMemcpyAsync(d_partitionList15, graph->h_partitionList + stream,
                        sizeof(*graph->h_partitionList), hipMemcpyHostToDevice,
                        streams[stream]);
        CalculateCostNSplitFrontiers11<<<staticGrid, blockDim, 0,
                                         streams[stream]>>>(
            d_partitionList15, graph->d_partitionsOffsets, graph->d_values,
            graph->d_frontier, d_filterEdges15, graph->d_offsets);
        break;
      }
      // GPUAssert(hipPeekAtLastError());

      //    hipMemcpyAsync(graph->d_partitionList + stream,
      //                      graph->h_partitionList + stream,
      //                      sizeof(*graph->h_partitionList),
      //                      hipMemcpyHostToDevice, streams[stream]);
      //
      // Touch the data afte using all 16 streams (256MB)
      // if (stream == 15) {
      //  uint32 numBlocks =
      //      (((*(graph->numVertices)) * WARP_SIZE + THREADS_PER_BLOCK) /
      //       THREADS_PER_BLOCK);
      //  dim3 gridDim(THREADS_PER_BLOCK,
      //               (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

      //  CalculateCostNSplitFrontiers14<<<gridDim, blockDim, 0,
      //                                   streams[stream]>>>(
      //      graph->d_partitionList, graph->d_partitionsOffsets,
      //      graph->d_values, graph->d_frontier, graph->d_filterEdges,
      //      graph->d_offsets);

      //  //  GPUAssert(hipPeekAtLastError());
      //}
    }

#endif
  }
  GPUAssert(hipDeviceSynchronize());

  totalProcess.endRecord();
  totalProcess.print();

  return;
}

//      if (*graph->frontierSize > 200000) {
//
//        // hipDeviceSynchronize();
//        CalculateCostNSplitFrontiers<uint32>
//            <<<staticGrid, blockDim, 0, partitionStream>>>(
//                graph->numPartitions, graph->d_partitionsOffsets,
//                graph->d_offsets, graph->d_zerocopyPartitionCost,
//                graph->d_demandFrontier, graph->d_filterFrontier);
//
//        CalculateCostNSplitFrontiers2<uint32>
//            <<<staticGrid, blockDim, 0, partitionStream>>>(
//                graph->numPartitions, graph->d_partitionsOffsets,
//                graph->d_offsets, graph->d_zerocopyPartitionCost,
//                graph->d_demandFrontier, graph->d_filterFrontier);
//
//        CalculateCostNSplitFrontiers3<uint32>
//            <<<staticGrid, blockDim, 0, partitionStream>>>(
//                graph->numPartitions, graph->d_partitionsOffsets,
//                graph->d_offsets, graph->d_zerocopyPartitionCost,
//                graph->d_demandFrontier, graph->d_filterFrontier);
//
//        hipMemcpyAsync(graph->h_partitionCost,
//        graph->d_zerocopyPartitionCost,
//                        *(graph->numPartitions) *
//                            sizeof(*graph->h_partitionCost),
//                        hipMemcpyDeviceToHost, partitionStream);
//
//        hipMemsetAsync(graph->d_zerocopyPartitionCost, 0,
//                        *(graph->numPartitions) *
//                            sizeof(*graph->h_partitionCost),
//                        partitionStream);
//
//        hipStreamSynchronize(partitionStream);
//      }
