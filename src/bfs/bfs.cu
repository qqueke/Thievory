#include "bfs.cuh"
#include <iostream>
#include <numa.h>
#include <ostream>
#include <queue>
#include <vector>

#define N_FILTER_STREAMS2 128

// Test the order between static and demand kernel
void BFS32(string filePath, uint32 srcVertex, double memAdvise, uint32 nRuns,
           uint32 nNeighborGPUs) {

  numa_run_on_node(0);
  ALGORITHM_TYPE algo = BFS;
  CSR<uint32> *graph = new CSR<uint32>;
  graph->ReadInputFile(filePath, algo);
  graph->InitData(srcVertex, nNeighborGPUs);
  // Adjust this number of blocks in x dimension to be a multiple of the number
  // of SMS and acquire better load balancing
  int device = 0; // Selected device
  uint32 k = 4;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint32 numSMs = prop.multiProcessorCount;

  dim3 staticGrid = dim3(k * numSMs, 1, 1);
  dim3 blockDim(THREADS_PER_BLOCK, 1, 1); // (x,y,z) = (THREADS_PER_BLOCK, 1, 1)

  hipStream_t staticStream, demandStream, frontierStream;

  GPUAssert(hipStreamCreate(&frontierStream));
  GPUAssert(hipStreamCreate(&staticStream));
  GPUAssert(hipStreamCreate(&demandStream));

  auto asyncFrontierPolicy = thrust::cuda::par_nosync.on(frontierStream);
  auto asyncStaticPolicy = thrust::cuda::par_nosync.on(staticStream);
  auto asyncDemandPolicy = thrust::cuda::par_nosync.on(demandStream);
  // auto syncPolicy  = thrust::cuda::par.on(staticStream);

  auto syncFrontierPolicy = thrust::cuda::par.on(frontierStream);
  auto syncStaticPolicy = thrust::cuda::par.on(staticStream);
  auto syncDemandPolicy = thrust::cuda::par.on(demandStream);

  TimeRecord<chrono::milliseconds> totalProcess("Total execution");
  TimeRecord<chrono::milliseconds> test0("Copy to GPU 0");
  TimeRecord<chrono::milliseconds> test1("Copy to GPU 1");
  TimeRecord<chrono::milliseconds> test2("Copy to GPU 2");
  TimeRecord<chrono::milliseconds> test3("Copy to GPU 3");

  TimeRecord<chrono::milliseconds> k0("Kernel GPU 0");
  TimeRecord<chrono::milliseconds> k1("Kernel GPU 1");
  TimeRecord<chrono::milliseconds> k2("Kernel GPU 2");
  TimeRecord<chrono::milliseconds> k3("Kernel GPU 3");

  uint32 nGPUs = nNeighborGPUs + 1;

  std::vector<std::array<hipStream_t, N_FILTER_STREAMS2>>
      neighborMemCpyStreams(nNeighborGPUs);

  std::vector<std::array<hipStream_t, N_FILTER_STREAMS2>>
      neighborComputeStreams(nNeighborGPUs);

  for (int i = 0; i < nNeighborGPUs; ++i) {
    hipSetDevice(i + 1);
    for (int j = 0; j < N_FILTER_STREAMS2; ++j)
      GPUAssert(hipStreamCreate(&neighborMemCpyStreams[i][j]));
  }

  hipSetDevice(0);

  for (int i = 0; i < nNeighborGPUs; ++i) {
    for (int j = 0; j < N_FILTER_STREAMS2; ++j)
      GPUAssert(hipStreamCreate(&neighborComputeStreams[i][j]));
  }

  hipStream_t streams[N_FILTER_STREAMS2];

  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++)
    GPUAssert(hipStreamCreate(&streams[i]));

  // Removing static data
  hipMemset(graph->d_inStatic, 0, *(graph->numVertices) * sizeof(bool));

  uint64 totalNumFilterPartitions = 0;
  std::cout << "Starting Traversals" << std::endl;
  for (int test = 0; test < nRuns; test++) {

    graph->ResetFrontierNValues();

    *(graph->frontierSize) = thrust::reduce(
        graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices), 0,
        thrust::plus<uint32>());

    totalProcess.startRecord();

    while (*(graph->frontierSize)) {

      setStaticNDemandFrontiers<<<staticGrid, blockDim, 0, frontierStream>>>(
          graph->numVertices, graph->d_frontier, graph->d_staticFrontier,
          graph->d_demandFrontier, graph->d_inStatic);

      hipStreamSynchronize(frontierStream);

      hipMemsetAsync(graph->d_frontier, 0,
                      *(graph->numVertices) * sizeof(*graph->d_frontier),
                      frontierStream);

      // Calculate the amount of active nodes in GPU memory
      *(graph->staticSize) =
          thrust::reduce(graph->thurstStaticFrontier,
                         graph->thurstStaticFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*graph->frontierSize > 10 * graph->avgVertPerPart) {
        CalculateActiveEdgesPerPartition<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        CalculateActiveEdgesRatio<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        SplitZeroCopyNFilterFrontiers<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        hipStreamSynchronize(demandStream);

        hipMemcpyAsync(graph->h_partitionCost, graph->d_partitionCost,
                        *graph->numPartitions * sizeof(*graph->h_partitionCost),
                        hipMemcpyDeviceToHost, streams[0]);

        hipMemsetAsync(graph->d_partitionCost, 0,
                        *graph->numPartitions * sizeof(*graph->d_partitionCost),
                        streams[0]);
      }

      // Calculate the amount of active vertices on-demand
      *(graph->demandSize) =
          thrust::reduce(graph->thurstDemandFrontier,
                         graph->thurstDemandFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*(graph->staticSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstStaticFrontier,
            graph->thurstStaticFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setStaticList<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->numVertices, graph->d_staticList, graph->d_staticFrontier,
            graph->d_prefixSum);

        hipStreamSynchronize(frontierStream);

        BFS32_Static_Kernel<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->staticSize, graph->d_staticList, graph->d_offsets,
            graph->d_staticEdges, graph->d_values, graph->d_frontier,
            graph->d_inStatic);
      }

      if (*(graph->demandSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstDemandFrontier,
            graph->thurstDemandFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setDemandList<<<staticGrid, blockDim, 0, demandStream>>>(
            graph->numVertices, graph->d_demandList, graph->d_demandFrontier,
            graph->d_prefixSum);

        uint32 numBlocks =
            (((*(graph->demandSize)) * WARP_SIZE + THREADS_PER_BLOCK) /
             THREADS_PER_BLOCK);
        dim3 gridDim(THREADS_PER_BLOCK,
                     (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

        hipStreamSynchronize(frontierStream);

        BFS32_Demand_Kernel<<<gridDim, blockDim, 0, demandStream>>>(
            graph->demandSize, graph->d_demandList, graph->d_values,
            graph->d_frontier, graph->h_edges, graph->d_offsets);
      }

      if (*graph->frontierSize > 10 * graph->avgVertPerPart) {
        uint32 numPartitionsOnTarget = 0;
        uint32 numPartitionsOnNeighbors = 0;

        std::queue<uint32> targetGPUQueue;
        std::vector<std::queue<uint32>> neighborGPUQueues(nNeighborGPUs);

        std::vector<uint32> partitionList;

        hipStreamSynchronize(streams[0]);

        for (uint32 partition = 0; partition < *graph->numPartitions;
             partition++) {

          if (graph->h_partitionCost[partition] <= FILTER_THRESHOLD)
            continue;

          partitionList.push_back(partition);
        }

        hipStreamSynchronize(frontierStream);

        totalNumFilterPartitions += partitionList.size();

        for (uint32 index = 0; index < partitionList.size(); index++) {

          uint32 partition = partitionList[index];

          // Partition edge start
          uint32 start =
              graph->h_offsets[graph->h_partitionsOffsets[partition]];

          uint32 partitionSize =
              graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
              start;

          // uint32 stream = partition % N_FILTER_STREAMS2;

          uint32 stream = (index / nGPUs) % N_FILTER_STREAMS2;

          graph->h_partitionList[stream] = partition;

          hipStreamSynchronize(streams[stream]);

          // hipDeviceSynchronize();
          test0.startRecord();
          hipMemcpyAsync(graph->d_filterEdges[stream], &graph->h_edges[start],
                          partitionSize * sizeof(*graph->h_edges),
                          hipMemcpyHostToDevice, streams[stream]);

          hipMemcpyAsync(&graph->d_partitionList[stream],
                          &graph->h_partitionList[stream],
                          sizeof(*graph->h_partitionList),
                          hipMemcpyHostToDevice, streams[stream]);

          //  hipDeviceSynchronize();
          test0.endRecord();

          targetGPUQueue.push(stream);

          for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {
            if (index + 1 >= partitionList.size())
              break;

            index++;
            partition = partitionList[index];

            // Partition edge start
            uint32 neighborStart =
                graph->h_offsets[graph->h_partitionsOffsets[partition]];

            uint32 neighborPartitionSize =
                graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
                neighborStart;

            uint32 neighborStream = (index / nGPUs) % N_FILTER_STREAMS2;

            graph->h_nPartList[gpu][neighborStream] = partition;

            // Sync compute stream on device 0
            hipStreamSynchronize(neighborComputeStreams[gpu][neighborStream]);
            // hipDeviceSynchronize();
            hipSetDevice(gpu + 1);

            //   hipDeviceSynchronize();
            test1.startRecord();

            hipMemcpyAsync(graph->d_nFilterEdges[gpu][neighborStream],
                            (gpu > 0) ? graph->h_edges2 + neighborStart
                                      : graph->h_edges + neighborStart,
                            neighborPartitionSize * sizeof(*graph->h_edges),
                            hipMemcpyHostToDevice,
                            neighborMemCpyStreams[gpu][neighborStream]);

            // We can prob allocate this data in the other numa node too
            hipMemcpyAsync(&graph->d_nPartList[gpu][neighborStream],
                            graph->h_nPartList[gpu] + neighborStream,
                            sizeof(**graph->h_nPartList),
                            hipMemcpyHostToDevice,
                            neighborMemCpyStreams[gpu][neighborStream]);

            //  hipDeviceSynchronize();
            test1.endRecord();

            neighborGPUQueues[gpu].push(neighborStream);

            hipSetDevice(0);
          }

          while (!targetGPUQueue.empty()) {
            uint32 tStream = targetGPUQueue.front();

            hipError_t streamStatus = hipStreamQuery(streams[tStream]);

            if (streamStatus == hipErrorNotReady) {
              if (targetGPUQueue.size() < N_FILTER_STREAMS2)
                continue;
              else
                hipStreamSynchronize(streams[tStream]);
            }

            targetGPUQueue.pop();
            numPartitionsOnTarget++;

            //   hipDeviceSynchronize();
            k0.startRecord();
            BFS32_Filter_Kernel<<<staticGrid, blockDim, 0, streams[tStream]>>>(
                &graph->d_partitionList[tStream], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_filterEdges[tStream], graph->d_offsets,
                graph->d_filterFrontier);
            //       hipDeviceSynchronize();
            k0.endRecord();
          }

          for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {

            while (!neighborGPUQueues[gpu].empty()) {
              uint32 nStream = neighborGPUQueues[gpu].front();

              hipSetDevice(gpu + 1);
              hipError_t streamStatus =
                  hipStreamQuery(neighborMemCpyStreams[gpu][nStream]);

              if (streamStatus == hipErrorNotReady) {
                if (neighborGPUQueues[gpu].size() < N_FILTER_STREAMS2) {
                  hipSetDevice(0);
                  continue;
                } else
                  hipStreamSynchronize(neighborMemCpyStreams[gpu][nStream]);
              }

              hipSetDevice(0);
              neighborGPUQueues[gpu].pop();
              numPartitionsOnNeighbors++;

              BFS32_NeighborFilter_Kernel<<<
                  staticGrid, blockDim, 0,
                  neighborComputeStreams[gpu][nStream]>>>(
                  &graph->d_nPartList[gpu][nStream], graph->d_partitionsOffsets,
                  graph->d_values, graph->d_frontier,
                  graph->d_nFilterEdges[gpu][nStream], graph->d_offsets,
                  graph->d_filterFrontier);
            }
          }
        }

        while (!targetGPUQueue.empty()) {

          uint32 tStream = targetGPUQueue.front();

          hipStreamSynchronize(streams[tStream]);
          targetGPUQueue.pop();
          numPartitionsOnTarget++;
          k0.startRecord();
          BFS32_Filter_Kernel<<<staticGrid, blockDim, 0, streams[tStream]>>>(
              &graph->d_partitionList[tStream], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier, graph->d_filterEdges[tStream],
              graph->d_offsets, graph->d_filterFrontier);
          //   hipDeviceSynchronize();
          k0.endRecord();
        }

        for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {

          while (!neighborGPUQueues[gpu].empty()) {
            uint32 nStream = neighborGPUQueues[gpu].front();

            hipSetDevice(gpu + 1);
            hipError_t streamStatus =
                hipStreamQuery(neighborMemCpyStreams[gpu][nStream]);

            if (streamStatus == hipErrorNotReady) {
              if (neighborGPUQueues[gpu].size() < N_FILTER_STREAMS2) {
                hipSetDevice(0);
                continue;
              } else
                hipStreamSynchronize(neighborMemCpyStreams[gpu][nStream]);
            }

            hipSetDevice(0);
            neighborGPUQueues[gpu].pop();
            numPartitionsOnNeighbors++;

            BFS32_NeighborFilter_Kernel<<<
                staticGrid, blockDim, 0,
                neighborComputeStreams[gpu][nStream]>>>(
                &graph->d_nPartList[gpu][nStream], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_nFilterEdges[gpu][nStream], graph->d_offsets,
                graph->d_filterFrontier);
          }
        }

        std::cout << "Partitions processed in target GPU: "
                  << numPartitionsOnTarget << std::endl;

        std::cout << "Partitions to be processed in neighbor GPUs: "
                  << numPartitionsOnNeighbors << std::endl;
      }

      hipDeviceSynchronize();

      *(graph->frontierSize) = thrust::reduce(
          graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices),
          0, thrust::plus<uint32>());
    }

    totalProcess.endRecord();
  }

  totalProcess.print();

  const uint64 partitionSizeMB = PARTITION_SIZE_MB / (1024 * 1024); // 1024^2

  uint64 MBytes = totalNumFilterPartitions * partitionSizeMB;

  // uint64 GBytes = MBytes >> 10;
  std::cout << "Total partitions in filter: " << totalNumFilterPartitions
            << std::endl;

  std::cout << "Total amount of data sent with filter: " << MBytes << " MB"
            << std::endl;

  graph->DumpValues();
  return;
}

void BFS64(string filePath, uint32 srcVertex, double memAdvise, uint32 nRuns) {

  return;
}
