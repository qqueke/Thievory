#include "bfs.cuh"
#include <iostream>
#include <numa.h>
#include <ostream>
#include <queue>
#include <vector>

#define N_FILTER_STREAMS2 128

// Test the order between static and demand kernel
void BFS32(string filePath, uint32 srcVertex, double memAdvise, uint32 nRuns,
           uint32 nNeighborGPUs) {

  numa_run_on_node(0);
  ALGORITHM_TYPE algo = BFS;
  CSR<uint32> *graph = new CSR<uint32>;
  graph->ReadInputFile(filePath, algo);
  graph->InitData(srcVertex, nNeighborGPUs);
  // Adjust this number of blocks in x dimension to be a multiple of the number
  // of SMS and acquire better load balancing
  int device = 0; // Selected device
  uint32 k = 4;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint32 numSMs = prop.multiProcessorCount;

  dim3 staticGrid = dim3(k * numSMs, 1, 1);
  dim3 blockDim(THREADS_PER_BLOCK, 1, 1); // (x,y,z) = (THREADS_PER_BLOCK, 1, 1)

  uint32 totalParts = 0;

  hipStream_t staticStream, demandStream, frontierStream;

  GPUAssert(hipStreamCreate(&frontierStream));
  GPUAssert(hipStreamCreate(&staticStream));
  GPUAssert(hipStreamCreate(&demandStream));

  auto asyncFrontierPolicy = thrust::cuda::par_nosync.on(frontierStream);
  auto asyncStaticPolicy = thrust::cuda::par_nosync.on(staticStream);
  auto asyncDemandPolicy = thrust::cuda::par_nosync.on(demandStream);
  // auto syncPolicy  = thrust::cuda::par.on(staticStream);

  auto syncFrontierPolicy = thrust::cuda::par.on(frontierStream);
  auto syncStaticPolicy = thrust::cuda::par.on(staticStream);
  auto syncDemandPolicy = thrust::cuda::par.on(demandStream);

  TimeRecord<chrono::milliseconds> totalProcess("Total execution");
  TimeRecord<chrono::milliseconds> test0("Copy to GPU 0");
  TimeRecord<chrono::milliseconds> test1("Copy to GPU 1");
  TimeRecord<chrono::milliseconds> test2("Copy to GPU 2");
  TimeRecord<chrono::milliseconds> test3("Copy to GPU 3");

  TimeRecord<chrono::milliseconds> k0("Kernel GPU 0");
  TimeRecord<chrono::milliseconds> k1("Kernel GPU 1");
  TimeRecord<chrono::milliseconds> k2("Kernel GPU 2");
  TimeRecord<chrono::milliseconds> k3("Kernel GPU 3");

  uint32 nGPUs = nNeighborGPUs + 1;

  std::vector<std::array<hipStream_t, N_FILTER_STREAMS2>>
      neighborMemCpyStreams(nNeighborGPUs);

  std::vector<std::array<hipStream_t, N_FILTER_STREAMS2>>
      neighborComputeStreams(nNeighborGPUs);

  for (int i = 0; i < nNeighborGPUs; ++i) {
    hipSetDevice(i + 1);
    for (int j = 0; j < N_FILTER_STREAMS2; ++j)
      GPUAssert(hipStreamCreate(&neighborMemCpyStreams[i][j]));
  }

  hipSetDevice(0);

  for (int i = 0; i < nNeighborGPUs; ++i) {
    for (int j = 0; j < N_FILTER_STREAMS2; ++j)
      GPUAssert(hipStreamCreate(&neighborComputeStreams[i][j]));
  }

  hipStream_t streams[N_FILTER_STREAMS2];

  for (uint32 i = 0; i < N_FILTER_STREAMS2; i++)
    GPUAssert(hipStreamCreate(&streams[i]));

  // Removing static data
  // hipMemset(graph->d_inStatic, 0, *(graph->numVertices) * sizeof(bool));

  GPUAssert(
      hipDeviceEnablePeerAccess(1, 0)); // Enable peer access with device 0

  GPUAssert(hipDeviceEnablePeerAccess(2, 0));

  GPUAssert(hipDeviceEnablePeerAccess(3, 0));

  graph->h_edges2 =
      (uint32 *)numa_alloc_onnode(graph->numEdges * sizeof(uint32), 1);

  hipHostRegister(graph->h_edges2, graph->numEdges * sizeof(uint32),
                   hipHostRegisterDefault);

  hipMemcpy(graph->h_edges2, graph->h_edges,
             graph->numEdges * sizeof(*graph->h_edges2), hipMemcpyHostToHost);

  hipDeviceSynchronize();

  std::cout << "Starting Traversals" << std::endl;
  for (int test = 0; test < nRuns; test++) {

    graph->ResetFrontierNValues();

    *(graph->frontierSize) = thrust::reduce(
        graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices), 0,
        thrust::plus<uint32>());

    totalProcess.startRecord();

    while (*(graph->frontierSize)) {

      setStaticNDemandFrontiers<<<staticGrid, blockDim, 0, frontierStream>>>(
          graph->numVertices, graph->d_frontier, graph->d_staticFrontier,
          graph->d_demandFrontier, graph->d_inStatic);

      hipStreamSynchronize(frontierStream);

      hipMemsetAsync(graph->d_frontier, 0,
                      *(graph->numVertices) * sizeof(*graph->d_frontier),
                      frontierStream);

      // Calculate the amount of active nodes in GPU memory
      *(graph->staticSize) =
          thrust::reduce(graph->thurstStaticFrontier,
                         graph->thurstStaticFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*graph->frontierSize > 1000000) {
        CalculateActiveEdgesPerPartition<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        CalculateActiveEdgesRatio<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        SplitZeroCopyNFilterFrontiers<uint32>
            <<<staticGrid, blockDim, 0, demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        hipStreamSynchronize(demandStream);

        hipMemcpyAsync(graph->h_partitionCost, graph->d_partitionCost,
                        *graph->numPartitions * sizeof(*graph->h_partitionCost),
                        hipMemcpyDeviceToHost, streams[0]);

        hipMemsetAsync(graph->d_partitionCost, 0,
                        *graph->numPartitions * sizeof(*graph->d_partitionCost),
                        streams[0]);
      }

      // Calculate the amount of active vertices on-demand
      *(graph->demandSize) =
          thrust::reduce(graph->thurstDemandFrontier,
                         graph->thurstDemandFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*(graph->staticSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstStaticFrontier,
            graph->thurstStaticFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setStaticList<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->numVertices, graph->d_staticList, graph->d_staticFrontier,
            graph->d_prefixSum);

        hipStreamSynchronize(frontierStream);

        BFS32_Static_Kernel<<<staticGrid, blockDim, 0, staticStream>>>(
            graph->staticSize, graph->d_staticList, graph->d_offsets,
            graph->d_staticEdges, graph->d_values, graph->d_frontier,
            graph->d_inStatic);
      }

      if (*(graph->demandSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstDemandFrontier,
            graph->thurstDemandFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setDemandList<<<staticGrid, blockDim, 0, demandStream>>>(
            graph->numVertices, graph->d_demandList, graph->d_demandFrontier,
            graph->d_prefixSum);

        uint32 numBlocks =
            (((*(graph->demandSize)) * WARP_SIZE + THREADS_PER_BLOCK) /
             THREADS_PER_BLOCK);
        dim3 gridDim(THREADS_PER_BLOCK,
                     (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

        hipStreamSynchronize(frontierStream);

        BFS32_Demand_Kernel<<<gridDim, blockDim, 0, demandStream>>>(
            graph->demandSize, graph->d_demandList, graph->d_values,
            graph->d_frontier, graph->h_edges, graph->d_offsets);
      }

      if (*graph->frontierSize > 1000000) {

        uint32 numParts = 0;
        uint32 numPartsNGPU = 0;

        uint32 numPartsNGPU0 = 0;
        uint32 numPartsNGPU1 = 0;
        uint32 numPartsNGPU2 = 0;
        uint32 numPartsNGPU3 = 0;

        uint32 teoNumParts = 0;

        std::queue<uint32> targetGPUQueue;
        std::vector<std::queue<uint32>> neighborGPUQueues(nNeighborGPUs);

        std::vector<uint32> partitionList;

        hipStreamSynchronize(streams[0]);

        for (uint32 partition = 0; partition < *graph->numPartitions;
             partition++) {

          if (graph->h_partitionCost[partition] <= FILTER_THRESHOLD)
            continue;

          partitionList.push_back(partition);
        }

        hipStreamSynchronize(frontierStream);

        for (uint32 index = 0; index < partitionList.size(); index++) {

          uint32 partition = partitionList[index];

          numParts++;

          // Partition edge start
          uint32 start =
              graph->h_offsets[graph->h_partitionsOffsets[partition]];

          uint32 partitionSize =
              graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
              start;

          // uint32 stream = partition % N_FILTER_STREAMS2;

          uint32 stream = (index / nGPUs) % N_FILTER_STREAMS2;

          graph->h_partitionList[stream] = partition;

          hipStreamSynchronize(streams[stream]);

          // hipDeviceSynchronize();
          test0.startRecord();
          hipMemcpyAsync(graph->d_filterEdges[stream], &graph->h_edges[start],
                          partitionSize * sizeof(*graph->h_edges),
                          hipMemcpyHostToDevice, streams[stream]);

          hipMemcpyAsync(&graph->d_partitionList[stream],
                          &graph->h_partitionList[stream],
                          sizeof(*graph->h_partitionList),
                          hipMemcpyHostToDevice, streams[stream]);

          //  hipDeviceSynchronize();
          test0.endRecord();

          targetGPUQueue.push(stream);

          for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {
            if (index + 1 >= partitionList.size())
              break;

            index++;
            partition = partitionList[index];

            teoNumParts++;

            // Partition edge start
            uint32 neighborStart =
                graph->h_offsets[graph->h_partitionsOffsets[partition]];

            uint32 neighborPartitionSize =
                graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
                neighborStart;

            uint32 neighborStream = (index / nGPUs) % N_FILTER_STREAMS2;

            graph->h_nPartList[gpu][neighborStream] = partition;

            // Sync compute stream on device 0
            hipStreamSynchronize(neighborComputeStreams[gpu][neighborStream]);
            // hipDeviceSynchronize();
            hipSetDevice(gpu + 1);

            //   hipDeviceSynchronize();
            test1.startRecord();

            hipMemcpyAsync(graph->d_nFilterEdges[gpu][neighborStream],
                            (gpu > 0) ? graph->h_edges2 + neighborStart
                                      : graph->h_edges + neighborStart,
                            neighborPartitionSize * sizeof(*graph->h_edges),
                            hipMemcpyHostToDevice,
                            neighborMemCpyStreams[gpu][neighborStream]);

            // We can prob allocate this data in the other numa node too
            hipMemcpyAsync(&graph->d_nPartList[gpu][neighborStream],
                            graph->h_nPartList[gpu] + neighborStream,
                            sizeof(*graph->h_neighborPartitionList),
                            hipMemcpyHostToDevice,
                            neighborMemCpyStreams[gpu][neighborStream]);

            //  hipDeviceSynchronize();
            test1.endRecord();

            neighborGPUQueues[gpu].push(neighborStream);

            hipSetDevice(0);
          }

          while (!targetGPUQueue.empty()) {
            uint32 tStream = targetGPUQueue.front();

            hipError_t streamStatus = hipStreamQuery(streams[tStream]);

            if (streamStatus == hipErrorNotReady) {
              if (targetGPUQueue.size() < N_FILTER_STREAMS2)
                continue;
              else
                hipStreamSynchronize(streams[tStream]);
            }

            numPartsNGPU0++;
            targetGPUQueue.pop();

            //   hipDeviceSynchronize();
            k0.startRecord();
            BFS32_Filter_Kernel<<<staticGrid, blockDim, 0, streams[tStream]>>>(
                &graph->d_partitionList[tStream], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_filterEdges[tStream], graph->d_offsets,
                graph->d_filterFrontier);
            //       hipDeviceSynchronize();
            k0.endRecord();
          }

          for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {

            while (!neighborGPUQueues[gpu].empty()) {
              uint32 nStream = neighborGPUQueues[gpu].front();

              hipSetDevice(gpu + 1);
              hipError_t streamStatus =
                  hipStreamQuery(neighborMemCpyStreams[gpu][nStream]);

              if (streamStatus == hipErrorNotReady) {
                if (neighborGPUQueues[gpu].size() < N_FILTER_STREAMS2) {
                  hipSetDevice(0);
                  continue;
                } else
                  hipStreamSynchronize(neighborMemCpyStreams[gpu][nStream]);
              }

              hipSetDevice(0);
              neighborGPUQueues[gpu].pop();

              BFS32_NeighborFilter_Kernel<<<
                  staticGrid, blockDim, 0,
                  neighborComputeStreams[gpu][nStream]>>>(
                  &graph->d_nPartList[gpu][nStream], graph->d_partitionsOffsets,
                  graph->d_values, graph->d_frontier,
                  graph->d_nFilterEdges[gpu][nStream], graph->d_offsets,
                  graph->d_filterFrontier);
            }
          }
        }

        while (!targetGPUQueue.empty()) {

          uint32 tStream = targetGPUQueue.front();

          hipStreamSynchronize(streams[tStream]);
          numPartsNGPU0++;
          targetGPUQueue.pop();

          k0.startRecord();
          BFS32_Filter_Kernel<<<staticGrid, blockDim, 0, streams[tStream]>>>(
              &graph->d_partitionList[tStream], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier, graph->d_filterEdges[tStream],
              graph->d_offsets, graph->d_filterFrontier);
          //   hipDeviceSynchronize();
          k0.endRecord();
        }

        for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {

          while (!neighborGPUQueues[gpu].empty()) {
            uint32 nStream = neighborGPUQueues[gpu].front();

            hipSetDevice(gpu + 1);
            hipError_t streamStatus =
                hipStreamQuery(neighborMemCpyStreams[gpu][nStream]);

            if (streamStatus == hipErrorNotReady) {
              if (neighborGPUQueues[gpu].size() < N_FILTER_STREAMS2) {
                hipSetDevice(0);
                continue;
              } else
                hipStreamSynchronize(neighborMemCpyStreams[gpu][nStream]);
            }

            hipSetDevice(0);
            neighborGPUQueues[gpu].pop();

            BFS32_NeighborFilter_Kernel<<<
                staticGrid, blockDim, 0,
                neighborComputeStreams[gpu][nStream]>>>(
                &graph->d_nPartList[gpu][nStream], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_nFilterEdges[gpu][nStream], graph->d_offsets,
                graph->d_filterFrontier);
            //   hipDeviceSynchronize();
          }
        }

        std::cout << "Partitions to be processed target GPU: " << numParts
                  << std::endl;
        std::cout << "Partitions processed in GPU 0: " << numPartsNGPU0
                  << std::endl;

        std::cout << "Partitions to be processed in neighbor GPUs: "
                  << teoNumParts << std::endl;

        std::cout << "Partitions processed in neighbor GPUs: " << numPartsNGPU
                  << std::endl;
        std::cout << "Partitions processed in neighbor GPU 1: " << numPartsNGPU1
                  << std::endl;
        std::cout << "Partitions processed in neighbor GPU 2: " << numPartsNGPU2
                  << std::endl;

        std::cout << "Partitions processed in neighbor GPU 3: " << numPartsNGPU3
                  << std::endl;

        totalParts += numParts + teoNumParts;
      }

      hipDeviceSynchronize();

      *(graph->frontierSize) = thrust::reduce(
          graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices),
          0, thrust::plus<uint32>());
    }
  }

  totalProcess.endRecord();
  totalProcess.print();

  // We're gonna need to compare results now!!
  hipMemcpy(graph->h_values, graph->d_values,
             *(graph->numVertices) * sizeof(uint32), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for (uint32 i = 0; i < 31; i++) {
    std::cout << "Our result: " << graph->h_values[i] << std::endl;
  }

  graph->DumpValues();
  return;
}

void BFS64(string filePath, uint32 srcVertex, double memAdvise, uint32 nRuns) {

  return;
}
