#include "bfs.cuh"
#include <iostream>
#include <numa.h>
#include <ostream>
#include <queue>
#include <vector>

void BFS32(std::string filePath, uint32 srcVertex, uint32 nRuns,
           uint32 nNeighborGPUs, std::unordered_map<int, int> affinityMap) {

  // numa_run_on_node(0);
  ALGORITHM_TYPE algo = BFS;
  CSR<uint32> *graph = new CSR<uint32>;
  graph->ReadInputFile(filePath, algo, srcVertex, nNeighborGPUs, affinityMap);
  graph->InitData();

  uint32 nGPUs = nNeighborGPUs + 1;
  // Adjust this number of blocks in x dimension to be a multiple of the number
  // of SMS and acquire better load balancing
  int device = 0; // Selected device
  uint32 k = 4;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  uint32 numSMs = prop.multiProcessorCount;

  dim3 staticGrid = dim3(k * numSMs, 1, 1);
  dim3 blockDim(THREADS_PER_BLOCK, 1, 1); // (x,y,z) = (THREADS_PER_BLOCK, 1, 1)

  // auto asyncFrontierPolicy =
  // thrust::cuda::par_nosync.on(graph->frontierStream); auto asyncStaticPolicy
  // = thrust::cuda::par_nosync.on(graph->staticStream); auto asyncDemandPolicy
  // = thrust::cuda::par_nosync.on(graph->demandStream); auto syncPolicy  =
  // thrust::cuda::par.on(staticStream);

  // auto syncFrontierPolicy = thrust::cuda::par.on(graph->frontierStream);
  // auto syncStaticPolicy = thrust::cuda::par.on(graph->staticStream);
  // auto syncDemandPolicy = thrust::cuda::par.on(graph->demandStream);

  uint64 totalNumFilterPartitions = 0;

  float totalDuration = 0.0f;

  graph->SetFrontierToRatio(1.0f);

  std::cout << "Starting Traversals" << std::endl;
  for (int test = 0; test < nRuns; test++) {

    graph->ResetFrontierNValues();

    *(graph->frontierSize) = thrust::reduce(
        graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices), 0,
        thrust::plus<uint32>());

    Timer timer("Execution time: ");
    while (*(graph->frontierSize)) {
      setStaticNDemandFrontiers<<<staticGrid, blockDim, 0,
                                  graph->frontierStream>>>(
          graph->numVertices, graph->d_frontier, graph->d_staticFrontier,
          graph->d_demandFrontier, graph->d_inStatic);

      hipStreamSynchronize(graph->frontierStream);

      hipMemsetAsync(graph->d_frontier, 0,
                      *(graph->numVertices) * sizeof(*graph->d_frontier),
                      graph->frontierStream);

      // Calculate the amount of active nodes in GPU memory
      *(graph->staticSize) =
          thrust::reduce(graph->thurstStaticFrontier,
                         graph->thurstStaticFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      // std::cout << "Static size: " << *graph->staticSize << std::endl;

      if (*graph->frontierSize > 10 * graph->avgVertPerPart) {
        CalculateActiveEdgesPerPartition<uint32>
            <<<staticGrid, blockDim, 0, graph->demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        CalculateActiveEdgesRatio<uint32>
            <<<staticGrid, blockDim, 0, graph->demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        SplitZeroCopyNFilterFrontiers<uint32>
            <<<staticGrid, blockDim, 0, graph->demandStream>>>(
                graph->numPartitions, graph->d_partitionsOffsets,
                graph->d_offsets, graph->d_partitionCost,
                graph->d_demandFrontier, graph->d_filterFrontier);

        hipStreamSynchronize(graph->demandStream);

        hipMemcpyAsync(graph->h_partitionCost, graph->d_partitionCost,
                        *graph->numPartitions * sizeof(*graph->h_partitionCost),
                        hipMemcpyDeviceToHost, graph->streams[0]);

        hipMemsetAsync(graph->d_partitionCost, 0,
                        *graph->numPartitions * sizeof(*graph->d_partitionCost),
                        graph->streams[0]);
      }

      // Calculate the amount of active vertices on-demand
      *(graph->demandSize) =
          thrust::reduce(graph->thurstDemandFrontier,
                         graph->thurstDemandFrontier + *(graph->numVertices), 0,
                         thrust::plus<uint32>());

      if (*(graph->staticSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstStaticFrontier,
            graph->thurstStaticFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setStaticList<<<staticGrid, blockDim, 0, graph->staticStream>>>(
            graph->numVertices, graph->d_staticList, graph->d_staticFrontier,
            graph->d_prefixSum);

        hipStreamSynchronize(graph->frontierStream);

        BFS32_Static_Kernel<<<staticGrid, blockDim, 0, graph->staticStream>>>(
            graph->staticSize, graph->d_staticList, graph->d_offsets,
            graph->d_staticEdges, graph->d_values, graph->d_frontier,
            graph->d_inStatic);
      }

      if (*(graph->demandSize) > 0) {

        thrust::exclusive_scan(
            graph->thurstDemandFrontier,
            graph->thurstDemandFrontier + *(graph->numVertices),
            graph->thurstPrefixSum, 0, thrust::plus<uint32>());

        setDemandList<<<staticGrid, blockDim, 0, graph->demandStream>>>(
            graph->numVertices, graph->d_demandList, graph->d_demandFrontier,
            graph->d_prefixSum);

        uint32 numBlocks =
            (((*(graph->demandSize)) * WARP_SIZE + THREADS_PER_BLOCK) /
             THREADS_PER_BLOCK);
        dim3 gridDim(THREADS_PER_BLOCK,
                     (numBlocks + THREADS_PER_BLOCK) / THREADS_PER_BLOCK);

        hipStreamSynchronize(graph->frontierStream);

        BFS32_Demand_Kernel<<<gridDim, blockDim, 0, graph->demandStream>>>(
            graph->demandSize, graph->d_demandList, graph->d_values,
            graph->d_frontier, graph->h_edges2[graph->GPUAffinityMap[0]],
            graph->d_offsets);
      }

      if (*graph->frontierSize > 10 * graph->avgVertPerPart) {
        uint32 numPartitionsOnTarget = 0;
        uint32 numPartitionsOnNeighbors = 0;

        std::queue<uint32> targetGPUQueue;
        std::vector<std::queue<uint32>> neighborGPUQueues(nNeighborGPUs);

        std::vector<uint32> partitionList;

        hipStreamSynchronize(graph->streams[0]);

        for (uint32 partition = 0; partition < *graph->numPartitions;
             partition++) {

          if (graph->h_partitionCost[partition] <= graph->h_filterThreshold)
            continue;

          partitionList.push_back(partition);
        }

        hipStreamSynchronize(graph->frontierStream);

        totalNumFilterPartitions += partitionList.size();

        for (uint32 index = 0; index < partitionList.size(); index++) {

          uint32 partition = partitionList[index];

          // Partition edge start
          uint32 start =
              graph->h_offsets[graph->h_partitionsOffsets[partition]];

          uint32 partitionSize =
              graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
              start;

          uint32 stream = (index / nGPUs) % N_TARGET_FILTER_STREAMS;

          graph->h_partitionList[stream] = partition;

          hipStreamSynchronize(graph->streams[stream]);

          // hipDeviceSynchronize();
          hipMemcpyAsync(graph->d_filterEdges[stream],
                          &graph->h_edges2[graph->GPUAffinityMap[0]][start],
                          partitionSize * sizeof(*graph->h_edges),
                          hipMemcpyHostToDevice, graph->streams[stream]);

          hipMemcpyAsync(&graph->d_partitionList[stream],
                          &graph->h_partitionList[stream],
                          sizeof(*graph->h_partitionList),
                          hipMemcpyHostToDevice, graph->streams[stream]);

          //  hipDeviceSynchronize();

          targetGPUQueue.push(stream);

          for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {
            if (index + 1 >= partitionList.size())
              break;

            index++;
            partition = partitionList[index];

            // Partition edge start
            uint32 neighborStart =
                graph->h_offsets[graph->h_partitionsOffsets[partition]];

            uint32 neighborPartitionSize =
                graph->h_offsets[graph->h_partitionsOffsets[partition + 1]] -
                neighborStart;

            uint32 neighborStream = (index / nGPUs) % N_FILTER_STREAMS;

            graph->h_nPartList[gpu][neighborStream] = partition;

            // Sync compute stream on device 0
            hipStreamSynchronize(
                graph->neighborComputeStreams[gpu][neighborStream]);
            // hipDeviceSynchronize();
            hipSetDevice(gpu + 1);

            //   hipDeviceSynchronize();

            hipMemcpyAsync(graph->d_nFilterEdges[gpu][neighborStream],
                            graph->h_edges2[graph->GPUAffinityMap[gpu + 1]] +
                                neighborStart,
                            neighborPartitionSize * sizeof(*graph->h_edges),
                            hipMemcpyHostToDevice,
                            graph->neighborMemCpyStreams[gpu][neighborStream]);

            // We can prob allocate this data in the other numa node too
            hipMemcpyAsync(&graph->d_nPartList[gpu][neighborStream],
                            graph->h_nPartList[gpu] + neighborStream,
                            sizeof(**graph->h_nPartList),
                            hipMemcpyHostToDevice,
                            graph->neighborMemCpyStreams[gpu][neighborStream]);

            //  hipDeviceSynchronize();

            neighborGPUQueues[gpu].push(neighborStream);

            hipSetDevice(0);
          }

          // while (!targetGPUQueue.empty())
          for (uint32 j = 0; j < targetGPUQueue.size(); j++) {
            uint32 tStream = targetGPUQueue.front();

            hipError_t streamStatus = hipStreamQuery(graph->streams[tStream]);

            if (streamStatus == hipErrorNotReady) {
              if (targetGPUQueue.size() < N_TARGET_FILTER_STREAMS)
                break;
              else
                hipStreamSynchronize(graph->streams[tStream]);
            }

            targetGPUQueue.pop();
            numPartitionsOnTarget++;

            //   hipDeviceSynchronize();
            BFS32_Filter_Kernel<<<staticGrid, blockDim, 0,
                                  graph->streams[tStream]>>>(
                &graph->d_partitionList[tStream], graph->d_partitionsOffsets,
                graph->d_values, graph->d_frontier,
                graph->d_filterEdges[tStream], graph->d_offsets,
                graph->d_filterFrontier);
            //       hipDeviceSynchronize();

            uint32 processedPartition = graph->h_partitionList[tStream];

            uint32 partitionStart =
                graph
                    ->h_offsets[graph->h_partitionsOffsets[processedPartition]];

            uint32 partitionEnd =
                graph->h_offsets[graph->h_partitionsOffsets[processedPartition +
                                                            1]];

            uint32 processedPartitionSize = partitionEnd - partitionStart;

            if (partitionEnd <= graph->numStaticEdges) {

              // hipStreamSynchronize(staticStreams[processedPartition]);
              hipMemcpyAsync(&graph->d_staticEdges[partitionStart],
                              graph->d_filterEdges[tStream],
                              processedPartitionSize * sizeof(*graph->h_edges),
                              hipMemcpyDeviceToDevice,
                              graph->streams[tStream]);

              hipMemsetAsync(
                  &graph->d_inStatic
                       [graph->h_partitionsOffsets[processedPartition]],
                  1,
                  (graph->h_partitionsOffsets[processedPartition + 1] -
                   graph->h_partitionsOffsets[processedPartition]) *
                      sizeof(*graph->d_inStatic),
                  graph->streams[tStream]);
              // hipDeviceSynchronize();
            }
          }

          for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {

            // while (!neighborGPUQueues[gpu].empty())
            for (uint32 j = 0; j < neighborGPUQueues[gpu].size(); j++) {
              uint32 nStream = neighborGPUQueues[gpu].front();

              hipSetDevice(gpu + 1);
              hipError_t streamStatus =
                  hipStreamQuery(graph->neighborMemCpyStreams[gpu][nStream]);

              if (streamStatus == hipErrorNotReady) {
                if (neighborGPUQueues[gpu].size() < N_FILTER_STREAMS) {
                  hipSetDevice(0);
                  break;
                } else
                  hipStreamSynchronize(
                      graph->neighborMemCpyStreams[gpu][nStream]);
              }

              hipSetDevice(0);
              neighborGPUQueues[gpu].pop();
              numPartitionsOnNeighbors++;
              uint32 processedPartition = graph->h_nPartList[gpu][nStream];

              uint32 partitionStart =
                  graph->h_offsets
                      [graph->h_partitionsOffsets[processedPartition]];

              uint32 partitionEnd =
                  graph->h_offsets
                      [graph->h_partitionsOffsets[processedPartition + 1]];

              uint32 processedPartitionSize = partitionEnd - partitionStart;

              if (partitionEnd <= graph->numStaticEdges) {

                // Aqui
                // hipSetDevice(gpu + 1);
                hipMemcpyAsync(&graph->d_staticEdges[partitionStart],
                                graph->d_nFilterEdges[gpu][nStream],
                                processedPartitionSize *
                                    sizeof(*graph->h_edges),
                                hipMemcpyDeviceToDevice,
                                graph->neighborComputeStreams[gpu][nStream]);

                // hipSetDevice(0);

                hipMemsetAsync(
                    &graph->d_inStatic
                         [graph->h_partitionsOffsets[processedPartition]],
                    1,
                    (graph->h_partitionsOffsets[processedPartition + 1] -
                     graph->h_partitionsOffsets[processedPartition]) *
                        sizeof(*graph->d_inStatic),
                    graph->neighborComputeStreams[gpu][nStream]);

                // hipDeviceSynchronize();
                BFS32_NeighborFilter_Kernel<<<
                    staticGrid, blockDim, 0,
                    graph->neighborComputeStreams[gpu][nStream]>>>(
                    &graph->d_nPartList[gpu][nStream],
                    graph->d_partitionsOffsets, graph->d_values,
                    graph->d_frontier, graph->d_staticEdges, graph->d_offsets,
                    graph->d_filterFrontier);
              } else {
                BFS32_NeighborFilter_Kernel<<<
                    staticGrid, blockDim, 0,
                    graph->neighborComputeStreams[gpu][nStream]>>>(
                    &graph->d_nPartList[gpu][nStream],
                    graph->d_partitionsOffsets, graph->d_values,
                    graph->d_frontier, graph->d_nFilterEdges[gpu][nStream],
                    graph->d_offsets, graph->d_filterFrontier);
              }
            }
          }
        }

        while (!targetGPUQueue.empty()) {

          uint32 tStream = targetGPUQueue.front();

          hipStreamSynchronize(graph->streams[tStream]);
          targetGPUQueue.pop();
          numPartitionsOnTarget++;
          BFS32_Filter_Kernel<<<staticGrid, blockDim, 0,
                                graph->streams[tStream]>>>(
              &graph->d_partitionList[tStream], graph->d_partitionsOffsets,
              graph->d_values, graph->d_frontier, graph->d_filterEdges[tStream],
              graph->d_offsets, graph->d_filterFrontier);
          //   hipDeviceSynchronize();

          uint32 processedPartition = graph->h_partitionList[tStream];

          uint32 partitionStart =
              graph->h_offsets[graph->h_partitionsOffsets[processedPartition]];

          uint32 partitionEnd =
              graph->h_offsets[graph->h_partitionsOffsets[processedPartition +
                                                          1]];

          uint32 processedPartitionSize = partitionEnd - partitionStart;

          if (partitionEnd <= graph->numStaticEdges) {

            // hipStreamSynchronize(staticStreams[processedPartition]);
            hipMemcpyAsync(&graph->d_staticEdges[partitionStart],
                            graph->d_filterEdges[tStream],
                            processedPartitionSize * sizeof(*graph->h_edges),
                            hipMemcpyDeviceToDevice, graph->streams[tStream]);

            hipMemsetAsync(
                &graph->d_inStatic
                     [graph->h_partitionsOffsets[processedPartition]],
                1,
                (graph->h_partitionsOffsets[processedPartition + 1] -
                 graph->h_partitionsOffsets[processedPartition]) *
                    sizeof(*graph->d_inStatic),
                graph->streams[tStream]);
            // hipDeviceSynchronize();
          }
        }

        for (uint32 gpu = 0; gpu < neighborGPUQueues.size(); gpu++) {

          while (!neighborGPUQueues[gpu].empty()) {
            uint32 nStream = neighborGPUQueues[gpu].front();

            hipSetDevice(gpu + 1);
            hipError_t streamStatus =
                hipStreamQuery(graph->neighborMemCpyStreams[gpu][nStream]);

            if (streamStatus == hipErrorNotReady) {
              if (neighborGPUQueues[gpu].size() < N_FILTER_STREAMS) {
                hipSetDevice(0);
                continue;
              } else
                hipStreamSynchronize(
                    graph->neighborMemCpyStreams[gpu][nStream]);
            }

            hipSetDevice(0);
            neighborGPUQueues[gpu].pop();
            numPartitionsOnNeighbors++;

            uint32 processedPartition = graph->h_nPartList[gpu][nStream];

            uint32 partitionStart =
                graph
                    ->h_offsets[graph->h_partitionsOffsets[processedPartition]];

            uint32 partitionEnd =
                graph->h_offsets[graph->h_partitionsOffsets[processedPartition +
                                                            1]];

            uint32 processedPartitionSize = partitionEnd - partitionStart;

            if (partitionEnd <= graph->numStaticEdges) {

              // Aqui
              // hipSetDevice(gpu + 1);
              hipMemcpyAsync(&graph->d_staticEdges[partitionStart],
                              graph->d_nFilterEdges[gpu][nStream],
                              processedPartitionSize * sizeof(*graph->h_edges),
                              hipMemcpyDeviceToDevice,
                              graph->neighborComputeStreams[gpu][nStream]);

              // hipSetDevice(0);

              hipMemsetAsync(
                  &graph->d_inStatic
                       [graph->h_partitionsOffsets[processedPartition]],
                  1,
                  (graph->h_partitionsOffsets[processedPartition + 1] -
                   graph->h_partitionsOffsets[processedPartition]) *
                      sizeof(*graph->d_inStatic),
                  graph->neighborComputeStreams[gpu][nStream]);

              // hipDeviceSynchronize();
              BFS32_NeighborFilter_Kernel<<<
                  staticGrid, blockDim, 0,
                  graph->neighborComputeStreams[gpu][nStream]>>>(
                  &graph->d_nPartList[gpu][nStream], graph->d_partitionsOffsets,
                  graph->d_values, graph->d_frontier, graph->d_staticEdges,
                  graph->d_offsets, graph->d_filterFrontier);
            } else {
              BFS32_NeighborFilter_Kernel<<<
                  staticGrid, blockDim, 0,
                  graph->neighborComputeStreams[gpu][nStream]>>>(
                  &graph->d_nPartList[gpu][nStream], graph->d_partitionsOffsets,
                  graph->d_values, graph->d_frontier,
                  graph->d_nFilterEdges[gpu][nStream], graph->d_offsets,
                  graph->d_filterFrontier);
            }
          }
        }

        // std::cout << "Partitions processed in target GPU: "
        //           << numPartitionsOnTarget << std::endl;
        //
        // std::cout << "Partitions to be processed in neighbor GPUs: "
        //           << numPartitionsOnNeighbors << std::endl;
      }

      hipDeviceSynchronize();

      *(graph->frontierSize) = thrust::reduce(
          graph->thrustFrontier, graph->thrustFrontier + *(graph->numVertices),
          0, thrust::plus<uint32>());
    }
    totalDuration += timer.GetDuration();
  }

  const uint64 partitionSizeMB = PARTITION_SIZE_MB / (1024 * 1024); // 1024^2

  uint64 MBytes = totalNumFilterPartitions * partitionSizeMB;

  std::cout << "Total amount of data sent with filter: " << MBytes << " MB"
            << std::endl;

  std::cout << "Average execution time: " << totalDuration / nRuns << " ms"
            << std::endl;

  graph->DumpValues();
  return;
}

void BFS64(std::string filePath, uint32 srcVertex, uint32 nRuns) { return; }
